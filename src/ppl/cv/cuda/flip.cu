#include "hip/hip_runtime.h"
/**
 * Licensed to the Apache Software Foundation (ASF) under one or more
 * contributor license agreements. See the NOTICE file distributed with this
 * work for additional information regarding copyright ownership. The ASF
 * licenses this file to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance with the
 * License. You may obtain a copy of the License at
 * http://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 */

#include "ppl/cv/cuda/flip.h"

#include "utility/utility.hpp"

using namespace ppl::common;

namespace ppl {
namespace cv {
namespace cuda {

template <typename T0, typename T1>
__global__
void flipKernel(const T1* src, int rows, int cols, int src_stride, T1* dst,
                int dst_stride, int flip_code) {
  int element_x = (blockIdx.x << kBlockShiftX1) + threadIdx.x;
  int element_y = (blockIdx.y << kBlockShiftY1) + threadIdx.y;
  T0* output = (T0*)((uchar*)dst + element_y * dst_stride);

  int x, y;
  if (flip_code == 0) {
    x = element_x;
    y = rows - element_y - 1;
  }
  else if (flip_code > 0) {
    x = cols - element_x - 1;
    y = element_y;
  }
  else {
    x = cols - element_x - 1;
    y = rows - element_y - 1;
  }
  T0* input = (T0*)((uchar*)src + y * src_stride);

  if (element_x < cols && element_y < rows) {
    T0 result = input[x];
    output[element_x] = result;
  }
}

RetCode flip(const uchar* src, int rows, int cols, int channels, int src_stride,
             uchar* dst, int dst_stride, int flip_code, hipStream_t stream) {
  PPL_ASSERT(src != nullptr);
  PPL_ASSERT(dst != nullptr);
  PPL_ASSERT(rows >= 1 && cols >= 1);
  PPL_ASSERT(channels == 1 || channels == 3 || channels == 4);
  PPL_ASSERT(src_stride >= cols * channels * (int)sizeof(uchar));
  PPL_ASSERT(dst_stride >= cols * channels * (int)sizeof(uchar));

  dim3 block, grid;
  block.x = kBlockDimX1;
  block.y = kBlockDimY1;
  grid.x  = divideUp(cols, kBlockDimX1, kBlockShiftX1);
  grid.y  = divideUp(rows, kBlockDimY1, kBlockShiftY1);

  if (channels == 1) {
    flipKernel<uchar, uchar><<<grid, block, 0, stream>>>(src, rows, cols,
        src_stride, dst, dst_stride, flip_code);
  }
  else if (channels == 3) {
    flipKernel<uchar3, uchar><<<grid, block, 0, stream>>>(src, rows, cols,
        src_stride, dst, dst_stride, flip_code);
  }
  else {  // channels == 4
    flipKernel<uchar4, uchar><<<grid, block, 0, stream>>>(src, rows, cols,
        src_stride, dst, dst_stride, flip_code);
  }

  hipError_t code = hipGetLastError();
  if (code != hipSuccess) {
    LOG(ERROR) << "CUDA error: " << hipGetErrorString(code);
    return RC_DEVICE_RUNTIME_ERROR;
  }

  return RC_SUCCESS;
}

RetCode flip(const float* src, int rows, int cols, int channels, int src_stride,
             float* dst, int dst_stride, int flip_code, hipStream_t stream) {
  PPL_ASSERT(src != nullptr);
  PPL_ASSERT(dst != nullptr);
  PPL_ASSERT(rows >= 1 && cols >= 1);
  PPL_ASSERT(channels == 1 || channels == 3 || channels == 4);
  PPL_ASSERT(src_stride >= cols * channels * (int)sizeof(float));
  PPL_ASSERT(dst_stride >= cols * channels * (int)sizeof(float));

  dim3 block, grid;
  block.x = kBlockDimX1;
  block.y = kBlockDimY1;
  grid.x  = divideUp(cols, kBlockDimX1, kBlockShiftX1);
  grid.y  = divideUp(rows, kBlockDimY1, kBlockShiftY1);

  if (channels == 1) {
    flipKernel<float, float><<<grid, block, 0, stream>>>(src, rows, cols,
        src_stride, dst, dst_stride, flip_code);
  }
  else if (channels == 3) {
    flipKernel<float3, float><<<grid, block, 0, stream>>>(src, rows, cols,
        src_stride, dst, dst_stride, flip_code);
  }
  else {  // channels == 4
    flipKernel<float4, float><<<grid, block, 0, stream>>>(src, rows, cols,
        src_stride, dst, dst_stride, flip_code);
  }

  hipError_t code = hipGetLastError();
  if (code != hipSuccess) {
    LOG(ERROR) << "CUDA error: " << hipGetErrorString(code);
    return RC_DEVICE_RUNTIME_ERROR;
  }

  return RC_SUCCESS;
}

template <>
RetCode Flip<uchar, 1>(hipStream_t stream,
                       int height,
                       int width,
                       int inWidthStride,
                       const uchar* inData,
                       int outWidthStride,
                       uchar* outData,
                       int flipCode) {
  RetCode code = flip(inData, height, width, 1, inWidthStride, outData,
                      outWidthStride, flipCode, stream);

  return code;
}

template <>
RetCode Flip<uchar, 3>(hipStream_t stream,
                       int height,
                       int width,
                       int inWidthStride,
                       const uchar* inData,
                       int outWidthStride,
                       uchar* outData,
                       int flipCode) {
  RetCode code = flip(inData, height, width, 3, inWidthStride, outData,
                      outWidthStride, flipCode, stream);

  return code;
}

template <>
RetCode Flip<uchar, 4>(hipStream_t stream,
                       int height,
                       int width,
                       int inWidthStride,
                       const uchar* inData,
                       int outWidthStride,
                       uchar* outData,
                       int flipCode) {
  RetCode code = flip(inData, height, width, 4, inWidthStride, outData,
                      outWidthStride, flipCode, stream);

  return code;
}

template <>
RetCode Flip<float, 1>(hipStream_t stream,
                       int height,
                       int width,
                       int inWidthStride,
                       const float* inData,
                       int outWidthStride,
                       float* outData,
                       int flipCode) {
  inWidthStride  *= sizeof(float);
  outWidthStride *= sizeof(float);
  RetCode code = flip(inData, height, width, 1, inWidthStride, outData,
                      outWidthStride, flipCode, stream);

  return code;
}

template <>
RetCode Flip<float, 3>(hipStream_t stream,
                       int height,
                       int width,
                       int inWidthStride,
                       const float* inData,
                       int outWidthStride,
                       float* outData,
                       int flipCode) {
  inWidthStride  *= sizeof(float);
  outWidthStride *= sizeof(float);
  RetCode code = flip(inData, height, width, 3, inWidthStride, outData,
                      outWidthStride, flipCode, stream);

  return code;
}

template <>
RetCode Flip<float, 4>(hipStream_t stream,
                       int height,
                       int width,
                       int inWidthStride,
                       const float* inData,
                       int outWidthStride,
                       float* outData,
                       int flipCode) {
  inWidthStride  *= sizeof(float);
  outWidthStride *= sizeof(float);
  RetCode code = flip(inData, height, width, 4, inWidthStride, outData,
                      outWidthStride, flipCode, stream);

  return code;
}

}  // namespace cuda
}  // namespace cv
}  // namespace ppl
