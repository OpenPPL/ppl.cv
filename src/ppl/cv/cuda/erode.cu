#include "hip/hip_runtime.h"
/**
 * Licensed to the Apache Software Foundation (ASF) under one or more
 * contributor license agreements. See the NOTICE file distributed with this
 * work for additional information regarding copyright ownership. The ASF
 * licenses this file to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance with the
 * License. You may obtain a copy of the License at
 * http://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 */

#include "ppl/cv/cuda/erode.h"
#include "morphology.hpp"

#include <cfloat>

#include "utility.hpp"

using namespace ppl::common;

namespace ppl {
namespace cv {
namespace cuda {

#define DEVICE_INLINE
#if defined(DEVICE_INLINE)
# define __DEVICE__ __device__ __forceinline__
#else
# define __DEVICE__ __device__
#endif

struct MinSwap {
  __DEVICE__
  void initialize(uchar &value0, uchar &value1, uchar &value2, uchar &value3) {
    value0 = 255;
    value1 = 255;
    value2 = 255;
    value3 = 255;
  }

  __DEVICE__
  void initialize(uchar &value) {
    value = 255;
  }

  __DEVICE__
  void initialize(uchar3 &value) {
    value.x = 255;
    value.y = 255;
    value.z = 255;
  }

  __DEVICE__
  void initialize(uchar4 &value) {
    value.x = 255;
    value.y = 255;
    value.z = 255;
    value.w = 255;
  }

  __DEVICE__
  void initialize(float &value) {
    value = FLT_MAX;
  }

  __DEVICE__
  void initialize(float3 &value) {
    value.x = FLT_MAX;
    value.y = FLT_MAX;
    value.z = FLT_MAX;
  }

  __DEVICE__
  void initialize(float4 &value) {
    value.x = FLT_MAX;
    value.y = FLT_MAX;
    value.z = FLT_MAX;
    value.w = FLT_MAX;
  }

  __DEVICE__
  void operator()(uchar &value, uchar &target) {
    value = value > target ? target : value;
  }

  __DEVICE__
  void operator()(uchar3 &value, uchar3 &target) {
    value.x = value.x > target.x ? target.x : value.x;
    value.y = value.y > target.y ? target.y : value.y;
    value.z = value.z > target.z ? target.z : value.z;
  }

  __DEVICE__
  void operator()(uchar4 &value, uchar4 &target) {
    value.x = value.x > target.x ? target.x : value.x;
    value.y = value.y > target.y ? target.y : value.y;
    value.z = value.z > target.z ? target.z : value.z;
    value.w = value.w > target.w ? target.w : value.w;
  }

  __DEVICE__
  void operator()(float &value, float &target) {
    value = value > target ? target : value;
  }

  __DEVICE__
  void operator()(float3 &value, float3 &target) {
    value.x = value.x > target.x ? target.x : value.x;
    value.y = value.y > target.y ? target.y : value.y;
    value.z = value.z > target.z ? target.z : value.z;
  }

  __DEVICE__
  void operator()(float4 &value, float4 &target) {
    value.x = value.x > target.x ? target.x : value.x;
    value.y = value.y > target.y ? target.y : value.y;
    value.z = value.z > target.z ? target.z : value.z;
    value.w = value.w > target.w ? target.w : value.w;
  }

  __DEVICE__
  void checkConstantResult(uchar &result, uchar border_value) {
    result = result > border_value ? border_value : result;
  }

  __DEVICE__
  void checkConstantResult(uchar3 &result, uchar border_value) {
    result.x = result.x > border_value ? border_value : result.x;
    result.y = 0;
    result.z = 0;
  }

  __DEVICE__
  void checkConstantResult(uchar4 &result, uchar border_value) {
    result.x = result.x > border_value ? border_value : result.x;
    result.y = 0;
    result.z = 0;
    result.w = 0;
  }

  __DEVICE__
  void checkConstantResult(float &result, float border_value) {
    result = result > border_value ? border_value : result;
  }

  __DEVICE__
  void checkConstantResult(float3 &result, float border_value) {
    result.x = result.x > border_value ? border_value : result.x;
    result.y = 0;
    result.z = 0;
  }

  __DEVICE__
  void checkConstantResult(float4 &result, float border_value) {
    result.x = result.x > border_value ? border_value : result.x;
    result.y = 0;
    result.z = 0;
    result.w = 0;
  }

  __DEVICE__
  void checkU8C1ConstantResult(uchar4 &result, uchar border_value,
                               bool constant_border0, bool constant_border1,
                               bool constant_border2, bool constant_border3) {
    if (constant_border0) {
      result.x = result.x > border_value ? border_value : result.x;
    }
    if (constant_border1) {
      result.y = result.y > border_value ? border_value : result.y;
    }
    if (constant_border2) {
      result.z = result.z > border_value ? border_value : result.z;
    }
    if (constant_border3) {
      result.w = result.w > border_value ? border_value : result.w;
    }
  }
};

RetCode erode(const uchar* src, int rows, int cols, int channels,
              int src_stride, uchar* dst, int dst_stride, const uchar* kernel,
              int kernel_y, int kernel_x, BorderType border_type,
              const uchar border_value, hipStream_t stream) {
  PPL_ASSERT(src != nullptr);
  PPL_ASSERT(dst != nullptr);
  PPL_ASSERT(rows > 0 && cols > 0);
  PPL_ASSERT(channels == 1 || channels == 3 || channels == 4);
  PPL_ASSERT(src_stride >= cols * channels * (int)sizeof(uchar));
  PPL_ASSERT(dst_stride >= cols * channels * (int)sizeof(uchar));
  PPL_ASSERT(kernel_y > 0 && kernel_y < rows);
  PPL_ASSERT(kernel_x > 0 && kernel_x < cols);
  PPL_ASSERT(kernel_y & 1 == 1 && kernel_x & 1 == 1);
  PPL_ASSERT(border_type == BORDER_TYPE_CONSTANT ||
             border_type == BORDER_TYPE_REPLICATE ||
             border_type == BORDER_TYPE_REFLECT ||
             border_type == BORDER_TYPE_WRAP ||
             border_type == BORDER_TYPE_REFLECT_101);

  hipError_t code;
  if (kernel_x == 1 && kernel_y == 1 && src_stride == dst_stride) {
    if (src != dst) {
      code = hipMemcpyAsync(dst, src, src_stride * rows,
                             hipMemcpyDeviceToDevice);
      if (code != hipSuccess) {
        LOG(ERROR) << "CUDA error: " << hipGetErrorString(code);
        return RC_DEVICE_MEMORY_ERROR;
      }
    }
    return RC_SUCCESS;
  }

  int diameter_x = kernel_x >> 1;
  int diameter_y = kernel_y >> 1;

  dim3 block, grid;
  block.x = kBlockDimX0;
  block.y = kBlockDimY0;
  grid.x  = divideUp(cols, kBlockDimX0, kBlockShiftX0);
  grid.y  = divideUp(rows, kBlockDimY0, kBlockShiftY0);

  bool all_masked = true;
  if (kernel != NULL) {
    int count = kernel_y * kernel_x;
    for (int index = 0; index < count; index++) {
      if (kernel[index] != 1) {
        all_masked = false;
        break;
      }
    }
  }

  MinSwap morphology_swap;

  if (all_masked) {
    uchar* buffer;
    size_t pitch;

    if (channels == 1) {
      int left_threads = divideUp(diameter_x, 4, 2);
      int remainders = cols & 3;
      remainders = remainders > diameter_x ? remainders : diameter_x;
      int aligned_columns = (cols - remainders) >> 2;
      int right_threads = cols - (aligned_columns << 2);
      int columns = aligned_columns + right_threads;

      if ((left_threads << 2) + right_threads <= cols) {
        dim3 block0, grid0;
        block0.x = kBlockDimX0;
        block0.y = kBlockDimY0;
        grid0.x  = divideUp(columns, kBlockDimX0, kBlockShiftX0);
        grid0.y  = divideUp(rows, kBlockDimY0, kBlockShiftY0);

        if (rows >= 480 && cols >= 640 && kernel_y >= 7 && kernel_x >= 7) {
          code = hipMallocPitch(&buffer, &pitch,
                                 cols * channels * sizeof(uchar), rows);
          if (code != hipSuccess) {
            LOG(ERROR) << "CUDA error: " << hipGetErrorString(code);
            return RC_DEVICE_MEMORY_ERROR;
          }
          morphRowU8C1Kernel0<MinSwap><<<grid0, block0, 0, stream>>>(src, rows,
              cols, columns, src_stride, left_threads, aligned_columns,
              diameter_x, buffer, pitch, morphology_swap);
          morphColKernel0<uchar, uchar, MinSwap><<<grid, block, 0, stream>>>(
              buffer, rows, cols, pitch, diameter_y, diameter_x, dst,
              dst_stride, border_type, border_value, morphology_swap);
          hipFree(buffer);
        }
        else {
          morph2DU8C1Kernel0<MinSwap><<<grid0, block0, 0, stream>>>(src, rows,
              cols, columns, src_stride, left_threads, aligned_columns,
              diameter_x, diameter_y, dst, dst_stride, border_type,
              border_value, morphology_swap);
        }
      }
      else {
        morph2DKernel0<uchar, uchar, MinSwap><<<grid, block, 0, stream>>>(src,
            rows, cols, src_stride, diameter_x, diameter_y, dst, dst_stride,
            border_type, border_value, morphology_swap);
      }
    }
    else if (channels == 3) {
      if (rows >= 480 && cols >= 640 && kernel_y >= 7 && kernel_x >= 7) {
        code = hipMallocPitch(&buffer, &pitch, cols * channels * sizeof(uchar),
                               rows);
        if (code != hipSuccess) {
          LOG(ERROR) << "CUDA error: " << hipGetErrorString(code);
          return RC_DEVICE_MEMORY_ERROR;
        }
        morphRowKernel0<uchar3, uchar, MinSwap><<<grid, block, 0, stream>>>(src,
            rows, cols, src_stride, diameter_x, buffer, pitch, morphology_swap);
        morphColKernel0<uchar3, uchar, MinSwap><<<grid, block, 0, stream>>>(
            buffer, rows, cols, pitch, diameter_y, diameter_x, dst, dst_stride,
            border_type, border_value, morphology_swap);
        hipFree(buffer);
      }
      else {
        morph2DKernel0<uchar3, uchar, MinSwap><<<grid, block, 0, stream>>>(src,
            rows, cols, src_stride, diameter_x, diameter_y, dst, dst_stride,
            border_type, border_value, morphology_swap);
      }
    }
    else {  // channels == 4
      if (rows >= 780 && cols >= 1024 && kernel_y >= 7 && kernel_x >= 7) {
        code = hipMallocPitch(&buffer, &pitch, cols * channels * sizeof(uchar),
                               rows);
        if (code != hipSuccess) {
          LOG(ERROR) << "CUDA error: " << hipGetErrorString(code);
          return RC_DEVICE_MEMORY_ERROR;
        }
        morphRowKernel0<uchar4, uchar, MinSwap><<<grid, block, 0, stream>>>(src,
            rows, cols, src_stride, diameter_x, buffer, pitch, morphology_swap);
        morphColKernel0<uchar4, uchar, MinSwap><<<grid, block, 0, stream>>>(
            buffer, rows, cols, pitch, diameter_y, diameter_x, dst, dst_stride,
            border_type, border_value, morphology_swap);
        hipFree(buffer);
      }
      else {
        morph2DKernel0<uchar4, uchar, MinSwap><<<grid, block, 0, stream>>>(src,
            rows, cols, src_stride, diameter_x, diameter_y, dst, dst_stride,
            border_type, border_value, morphology_swap);
      }
    }
  }
  else {
    uchar* mask;
    int size = kernel_y * kernel_x * sizeof(uchar);
    code = hipMalloc(&mask, size);
    if (code != hipSuccess) {
      LOG(ERROR) << "CUDA error: " << hipGetErrorString(code);
      return RC_DEVICE_MEMORY_ERROR;
    }
    code = hipMemcpyAsync(mask, kernel, size, hipMemcpyHostToDevice);
    if (code != hipSuccess) {
      hipFree(mask);
      LOG(ERROR) << "CUDA error: " << hipGetErrorString(code);
      return RC_DEVICE_MEMORY_ERROR;
    }

    if (channels == 1) {
      int left_threads = divideUp(diameter_x, 4, 2);
      int remainders = cols & 3;
      remainders = remainders > diameter_x ? remainders : diameter_x;
      int aligned_columns = (cols - remainders) >> 2;
      int right_threads = cols - (aligned_columns << 2);
      int columns = aligned_columns + right_threads;

      if ((left_threads << 2) + right_threads <= cols) {
        dim3 block0, grid0;
        block0.x = kBlockDimX0;
        block0.y = kBlockDimY0;
        grid0.x  = divideUp(columns, kBlockDimX0, kBlockShiftX0);
        grid0.y  = divideUp(rows, kBlockDimY0, kBlockShiftY0);

        morph2DU8C1Kernel1<MinSwap><<<grid0, block0, 0, stream>>>(src, rows,
            cols, columns, src_stride, mask, left_threads, aligned_columns,
            diameter_x, diameter_y, kernel_x, kernel_y, dst, dst_stride,
            border_type, border_value, morphology_swap);
      }
      else {
        morph2DKernel1<uchar, uchar, MinSwap><<<grid, block, 0, stream>>>(src,
            rows, cols, src_stride, mask, diameter_x, diameter_y, kernel_x,
            kernel_y, dst, dst_stride, border_type, border_value,
            morphology_swap);
      }
    }
    else if (channels == 3) {
      morph2DKernel1<uchar3, uchar, MinSwap><<<grid, block, 0, stream>>>(src,
          rows, cols, src_stride, mask, diameter_x, diameter_y, kernel_x,
          kernel_y, dst, dst_stride, border_type, border_value,
          morphology_swap);
    }
    else {
      morph2DKernel1<uchar4, uchar, MinSwap><<<grid, block, 0, stream>>>(src,
          rows, cols, src_stride, mask, diameter_x, diameter_y, kernel_x,
          kernel_y, dst, dst_stride, border_type, border_value,
          morphology_swap);
    }
    hipFree(mask);
  }

  return RC_SUCCESS;
}

RetCode erode(const float* src, int rows, int cols, int channels,
              int src_stride, float* dst, int dst_stride, const uchar* kernel,
              int kernel_y, int kernel_x, BorderType border_type,
              const float border_value, hipStream_t stream) {
  PPL_ASSERT(src != nullptr);
  PPL_ASSERT(dst != nullptr);
  PPL_ASSERT(rows > 0 && cols > 0);
  PPL_ASSERT(channels == 1 || channels == 3 || channels == 4);
  PPL_ASSERT(src_stride >= cols * channels * (int)sizeof(float));
  PPL_ASSERT(dst_stride >= cols * channels * (int)sizeof(float));
  PPL_ASSERT(kernel_y > 0 && kernel_y < rows);
  PPL_ASSERT(kernel_x > 0 && kernel_x < cols);
  PPL_ASSERT(border_type == BORDER_TYPE_CONSTANT ||
             border_type == BORDER_TYPE_REPLICATE ||
             border_type == BORDER_TYPE_REFLECT ||
             border_type == BORDER_TYPE_WRAP ||
             border_type == BORDER_TYPE_REFLECT_101);

  hipError_t code;
  if (kernel_x == 1 && kernel_y == 1 && src_stride == dst_stride) {
    if (src != dst) {
      code = hipMemcpyAsync(dst, src, src_stride * rows,
                             hipMemcpyDeviceToDevice);
      if (code != hipSuccess) {
        LOG(ERROR) << "CUDA error: " << hipGetErrorString(code);
        return RC_DEVICE_MEMORY_ERROR;
      }
    }
    return RC_SUCCESS;
  }

  int diameter_x = kernel_x >> 1;
  int diameter_y = kernel_y >> 1;

  dim3 block, grid;
  block.x = kBlockDimX1;
  block.y = kBlockDimY1;
  grid.x  = divideUp(cols, kBlockDimX1, kBlockShiftX1);
  grid.y  = divideUp(rows, kBlockDimY1, kBlockShiftY1);

  bool all_masked = true;
  if (kernel != NULL) {
    int count = kernel_y * kernel_x;
    for (int index = 0; index < count; index++) {
      if (kernel[index] != 1) {
        all_masked = false;
        break;
      }
    }
  }

  MinSwap morphology_swap;

  if (all_masked) {
    float* buffer;
    size_t pitch;

    if (channels == 1) {
      if (rows >= 480 && cols >= 640 && kernel_y >= 7 && kernel_x >= 7) {
        code = hipMallocPitch(&buffer, &pitch, cols * channels * sizeof(float),
                               rows);
        if (code != hipSuccess) {
          LOG(ERROR) << "CUDA error: " << hipGetErrorString(code);
          return RC_DEVICE_MEMORY_ERROR;
        }
        morphRowKernel0<float, float, MinSwap><<<grid, block, 0, stream>>>(src,
            rows, cols, src_stride, diameter_x, buffer, pitch, morphology_swap);
        morphColKernel0<float, float, MinSwap><<<grid, block, 0, stream>>>(
            buffer, rows, cols, pitch, diameter_y, diameter_x, dst, dst_stride,
            border_type, border_value, morphology_swap);
        hipFree(buffer);
      }
      else {
        morph2DKernel0<float, float, MinSwap><<<grid, block, 0, stream>>>(src,
            rows, cols, src_stride, diameter_x, diameter_y, dst, dst_stride,
            border_type, border_value, morphology_swap);
      }
    }
    else if (channels == 3) {
      if (rows >= 480 && cols >= 640 && kernel_y >= 7 && kernel_x >= 7) {
        code = hipMallocPitch(&buffer, &pitch, cols * channels * sizeof(float),
                               rows);
        if (code != hipSuccess) {
          LOG(ERROR) << "CUDA error: " << hipGetErrorString(code);
          return RC_DEVICE_MEMORY_ERROR;
        }
        morphRowKernel0<float3, float, MinSwap><<<grid, block, 0, stream>>>(src,
            rows, cols, src_stride, diameter_x, buffer, pitch, morphology_swap);
        morphColKernel0<float3, float, MinSwap><<<grid, block, 0, stream>>>(
            buffer, rows, cols, pitch, diameter_y, diameter_x, dst, dst_stride,
            border_type, border_value, morphology_swap);
        hipFree(buffer);
      }
      else {
        morph2DKernel0<float3, float, MinSwap><<<grid, block, 0, stream>>>(src,
            rows, cols, src_stride, diameter_x, diameter_y, dst, dst_stride,
            border_type, border_value, morphology_swap);
      }
    }
    else {  // channels == 4
      if (rows >= 480 && cols >= 640 && kernel_y >= 7 && kernel_x >= 7) {
        code = hipMallocPitch(&buffer, &pitch, cols * channels * sizeof(float),
                               rows);
        if (code != hipSuccess) {
          LOG(ERROR) << "CUDA error: " << hipGetErrorString(code);
          return RC_DEVICE_MEMORY_ERROR;
        }
        morphRowKernel0<float4, float, MinSwap><<<grid, block, 0, stream>>>(src,
            rows, cols, src_stride, diameter_x, buffer, pitch, morphology_swap);
        morphColKernel0<float4, float, MinSwap><<<grid, block, 0, stream>>>(
            buffer, rows, cols, pitch, diameter_y, diameter_x, dst, dst_stride,
            border_type, border_value, morphology_swap);
        hipFree(buffer);
      }
      else {
        morph2DKernel0<float4, float, MinSwap><<<grid, block, 0, stream>>>(src,
            rows, cols, src_stride, diameter_x, diameter_y, dst, dst_stride,
            border_type, border_value, morphology_swap);
      }
    }
  }
  else {
    uchar* mask;
    int size = kernel_y * kernel_x * sizeof(uchar);
    code = hipMalloc(&mask, size);
    if (code != hipSuccess) {
      LOG(ERROR) << "CUDA error: " << hipGetErrorString(code);
      return RC_DEVICE_MEMORY_ERROR;
    }
    code = hipMemcpyAsync(mask, kernel, size, hipMemcpyHostToDevice);
    if (code != hipSuccess) {
      hipFree(mask);
      LOG(ERROR) << "CUDA error: " << hipGetErrorString(code);
      return RC_DEVICE_MEMORY_ERROR;
    }
    if (channels == 1) {
      morph2DKernel1<float, float, MinSwap><<<grid, block, 0, stream>>>(src,
          rows, cols, src_stride, mask, diameter_x, diameter_y, kernel_x,
          kernel_y, dst, dst_stride, border_type, border_value,
          morphology_swap);
    }
    else if (channels == 3) {
      morph2DKernel1<float3, float, MinSwap><<<grid, block, 0, stream>>>(src,
          rows, cols, src_stride, mask, diameter_x, diameter_y, kernel_x,
          kernel_y, dst, dst_stride, border_type, border_value,
          morphology_swap);
    }
    else {
      morph2DKernel1<float4, float, MinSwap><<<grid, block, 0, stream>>>(src,
          rows, cols, src_stride, mask, diameter_x, diameter_y, kernel_x,
          kernel_y, dst, dst_stride, border_type, border_value,
          morphology_swap);
    }
    hipFree(mask);
  }

  return RC_SUCCESS;
}

template <>
RetCode Erode<uchar, 1>(hipStream_t stream,
                        int height,
                        int width,
                        int inWidthStride,
                        const uchar* inData,
                        int kernelx_len,
                        int kernely_len,
                        const uchar* kernel,
                        int outWidthStride,
                        uchar* outData,
                        BorderType border_type,
                        const uchar border_value) {
  RetCode code = erode(inData, height, width, 1, inWidthStride, outData,
                       outWidthStride, kernel, kernely_len, kernelx_len,
                       border_type, border_value, stream);

  return code;
}

template <>
RetCode Erode<uchar, 3>(hipStream_t stream,
                        int height,
                        int width,
                        int inWidthStride,
                        const uchar* inData,
                        int kernelx_len,
                        int kernely_len,
                        const uchar* kernel,
                        int outWidthStride,
                        uchar* outData,
                        BorderType border_type,
                        const uchar border_value) {
  RetCode code = erode(inData, height, width, 3, inWidthStride, outData,
                       outWidthStride, kernel, kernely_len, kernelx_len,
                       border_type, border_value, stream);

  return code;
}

template <>
RetCode Erode<uchar, 4>(hipStream_t stream,
                        int height,
                        int width,
                        int inWidthStride,
                        const uchar* inData,
                        int kernelx_len,
                        int kernely_len,
                        const uchar* kernel,
                        int outWidthStride,
                        uchar* outData,
                        BorderType border_type,
                        const uchar border_value) {
  RetCode code = erode(inData, height, width, 4, inWidthStride, outData,
                       outWidthStride, kernel, kernely_len, kernelx_len,
                       border_type, border_value, stream);

  return code;
}

template <>
RetCode Erode<float, 1>(hipStream_t stream,
                        int height,
                        int width,
                        int inWidthStride,
                        const float* inData,
                        int kernelx_len,
                        int kernely_len,
                        const uchar* kernel,
                        int outWidthStride,
                        float* outData,
                        BorderType border_type,
                        const float border_value) {
  inWidthStride  *= sizeof(float);
  outWidthStride *= sizeof(float);
  RetCode code = erode(inData, height, width, 1, inWidthStride, outData,
                       outWidthStride, kernel, kernely_len, kernelx_len,
                       border_type, border_value, stream);

  return code;
}

template <>
RetCode Erode<float, 3>(hipStream_t stream,
                        int height,
                        int width,
                        int inWidthStride,
                        const float* inData,
                        int kernelx_len,
                        int kernely_len,
                        const uchar* kernel,
                        int outWidthStride,
                        float* outData,
                        BorderType border_type,
                        const float border_value) {
  inWidthStride  *= sizeof(float);
  outWidthStride *= sizeof(float);
  RetCode code = erode(inData, height, width, 3, inWidthStride, outData,
                       outWidthStride, kernel, kernely_len, kernelx_len,
                       border_type, border_value, stream);

  return code;
}

template <>
RetCode Erode<float, 4>(hipStream_t stream,
                        int height,
                        int width,
                        int inWidthStride,
                        const float* inData,
                        int kernelx_len,
                        int kernely_len,
                        const uchar* kernel,
                        int outWidthStride,
                        float* outData,
                        BorderType border_type,
                        const float border_value) {
  inWidthStride  *= sizeof(float);
  outWidthStride *= sizeof(float);
  RetCode code = erode(inData, height, width, 4, inWidthStride, outData,
                       outWidthStride, kernel, kernely_len, kernelx_len,
                       border_type, border_value, stream);

  return code;
}

}  // namespace cuda
}  // namespace cv
}  // namespace ppl
