#include "hip/hip_runtime.h"
/**
 * Licensed to the Apache Software Foundation (ASF) under one or more
 * contributor license agreements. See the NOTICE file distributed with this
 * work for additional information regarding copyright ownership. The ASF
 * licenses this file to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance with the
 * License. You may obtain a copy of the License at
 * http://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 */

#include "ppl/cv/cuda/crop.h"

#include "utility.hpp"

using namespace ppl::common;

namespace ppl {
namespace cv {
namespace cuda {

template <typename T>
__global__
void cropKernel(const T* src, int src_stride, const int top, const int left,
                const float scale, T* dst, int dst_rows, int dst_cols,
                int dst_stride) {
  int element_x = ((blockIdx.x << kBlockShiftX1) + threadIdx.x) << 2;
  int element_y = (blockIdx.y << kBlockShiftY1) + threadIdx.y;
  if (element_x >= dst_cols || element_y >= dst_rows) {
    return;
  }

  T* input = (T*)((uchar*)src + (top + element_y) * src_stride);
  T value0, value1, value2, value3;
  value0 = input[left + element_x];
  value1 = input[left + element_x + 1];
  value2 = input[left + element_x + 2];
  value3 = input[left + element_x + 3];

  if (scale != 1.f) {
    float fvalue0, fvalue1, fvalue2, fvalue3;
    fvalue0 = value0 * scale;
    fvalue1 = value1 * scale;
    fvalue2 = value2 * scale;
    fvalue3 = value3 * scale;

    if (sizeof(T) == 1) {
      value0 = saturate_cast(fvalue0);
      value1 = saturate_cast(fvalue1);
      value2 = saturate_cast(fvalue2);
      value3 = saturate_cast(fvalue3);
    }
    else {
      value0 = fvalue0;
      value1 = fvalue1;
      value2 = fvalue2;
      value3 = fvalue3;
    }
  }

  T* output = (T*)((uchar*)dst + element_y * dst_stride);
  if (element_x < dst_cols - 3) {
    output[element_x] = value0;
    output[element_x + 1] = value1;
    output[element_x + 2] = value2;
    output[element_x + 3] = value3;
  }
  else {
    output[element_x] = value0;
    if (element_x < dst_cols - 1) {
      output[element_x + 1] = value1;
    }
    if (element_x < dst_cols - 2) {
      output[element_x + 2] = value2;
    }
  }
}

RetCode crop(const uchar* src, int src_rows, int src_cols, int channels,
             int src_stride, uchar* dst, int dst_rows, int dst_cols,
             int dst_stride, const int left, const int top, const float scale,
             hipStream_t stream) {
  PPL_ASSERT(src != nullptr);
  PPL_ASSERT(dst != nullptr);
  PPL_ASSERT(src_rows >= 1 && src_cols >= 1);
  PPL_ASSERT(dst_rows >= 1 && dst_cols >= 1);
  PPL_ASSERT(src_rows >= dst_rows && src_cols >= dst_cols);
  PPL_ASSERT(left >= 0 && left < src_cols);
  PPL_ASSERT(top >= 0 && top < src_rows);
  PPL_ASSERT(channels == 1 || channels == 3 || channels == 4);
  PPL_ASSERT(src_stride >= src_cols * channels * (int)sizeof(uchar));
  PPL_ASSERT(dst_stride >= dst_cols * channels * (int)sizeof(uchar));

  int columns = dst_cols * channels;
  dim3 block, grid;
  block.x = kBlockDimX1;
  block.y = kBlockDimY1;
  grid.x  = divideUp(divideUp(columns, 4, 2), kBlockDimX1, kBlockShiftX1);
  grid.y  = divideUp(dst_rows, kBlockDimY1, kBlockShiftY1);

  hipError_t code;
  if (scale == 1.f) {
    uchar* src_start = (uchar*)src + top * src_stride +
                       left * channels * sizeof(uchar);
    code = hipMemcpy2D(dst, dst_stride, src_start, src_stride,
                        dst_cols * channels * sizeof(uchar), dst_rows,
                        hipMemcpyDeviceToDevice);
    if (code != hipSuccess) {
      LOG(ERROR) << "CUDA error: " << hipGetErrorString(code);
      return RC_DEVICE_MEMORY_ERROR;
    }
  }
  else {
    cropKernel<uchar><<<grid, block, 0, stream>>>(src, src_stride, top,
        left * channels, scale, dst, dst_rows, columns, dst_stride);
    code = hipGetLastError();
    if (code != hipSuccess) {
      LOG(ERROR) << "CUDA error: " << hipGetErrorString(code);
      return RC_DEVICE_RUNTIME_ERROR;
    }
  }

  return RC_SUCCESS;
}

RetCode crop(const float* src, int src_rows, int src_cols, int channels,
             int src_stride, float* dst, int dst_rows, int dst_cols,
             int dst_stride, const int left, const int top, const float scale,
             hipStream_t stream) {
  PPL_ASSERT(src != nullptr);
  PPL_ASSERT(dst != nullptr);
  PPL_ASSERT(src_rows >= 1 && src_cols >= 1);
  PPL_ASSERT(dst_rows >= 1 && dst_cols >= 1);
  PPL_ASSERT(src_rows >= dst_rows && src_cols >= dst_cols);
  PPL_ASSERT(left >= 0 && left < src_cols);
  PPL_ASSERT(top >= 0 && top < src_rows);
  PPL_ASSERT(channels == 1 || channels == 3 || channels == 4);
  PPL_ASSERT(src_stride >= src_cols * channels * (int)sizeof(float));
  PPL_ASSERT(dst_stride >= dst_cols * channels * (int)sizeof(float));

  int columns = dst_cols * channels;
  dim3 block, grid;
  block.x = kBlockDimX1;
  block.y = kBlockDimY1;
  grid.x  = divideUp(divideUp(columns, 4, 2), kBlockDimX1, kBlockShiftX1);
  grid.y  = divideUp(dst_rows, kBlockDimY1, kBlockShiftY1);

  hipError_t code;
  if (scale == 1.f) {
    float* src_start = (float*)((uchar*)src + top * src_stride +
                       left * channels * sizeof(float));
    code = hipMemcpy2D(dst, dst_stride, src_start, src_stride,
                        dst_cols * channels * sizeof(float), dst_rows,
                        hipMemcpyDeviceToDevice);
    if (code != hipSuccess) {
      LOG(ERROR) << "CUDA error: " << hipGetErrorString(code);
      return RC_DEVICE_MEMORY_ERROR;
    }
  }
  else {
    cropKernel<float><<<grid, block, 0, stream>>>(src, src_stride, top,
        left * channels, scale, dst, dst_rows, columns, dst_stride);
    code = hipGetLastError();
    if (code != hipSuccess) {
      LOG(ERROR) << "CUDA error: " << hipGetErrorString(code);
      return RC_DEVICE_RUNTIME_ERROR;
    }
  }

  return RC_SUCCESS;
}

template <>
RetCode Crop<uchar, 1>(hipStream_t stream,
                       int inHeight,
                       int inWidth,
                       int inWidthStride,
                       const uchar* inData,
                       int outHeight,
                       int outWidth,
                       int outWidthStride,
                       uchar* outData,
                       const int left,
                       const int top,
                       const float scale) {
  RetCode code = crop(inData, inHeight, inWidth, 1, inWidthStride, outData,
                      outHeight, outWidth, outWidthStride, left, top, scale,
                      stream);

  return code;
}

template <>
RetCode Crop<uchar, 3>(hipStream_t stream,
                       int inHeight,
                       int inWidth,
                       int inWidthStride,
                       const uchar* inData,
                       int outHeight,
                       int outWidth,
                       int outWidthStride,
                       uchar* outData,
                       const int left,
                       const int top,
                       const float scale) {
  RetCode code = crop(inData, inHeight, inWidth, 3, inWidthStride, outData,
                      outHeight, outWidth, outWidthStride, left, top, scale,
                      stream);

  return code;
}

template <>
RetCode Crop<uchar, 4>(hipStream_t stream,
                       int inHeight,
                       int inWidth,
                       int inWidthStride,
                       const uchar* inData,
                       int outHeight,
                       int outWidth,
                       int outWidthStride,
                       uchar* outData,
                       const int left,
                       const int top,
                       const float scale) {
  RetCode code = crop(inData, inHeight, inWidth, 4, inWidthStride, outData,
                      outHeight, outWidth, outWidthStride, left, top, scale,
                      stream);

  return code;
}

template <>
RetCode Crop<float, 1>(hipStream_t stream,
                       int inHeight,
                       int inWidth,
                       int inWidthStride,
                       const float* inData,
                       int outHeight,
                       int outWidth,
                       int outWidthStride,
                       float* outData,
                       const int left,
                       const int top,
                       const float scale) {
  inWidthStride  *= sizeof(float);
  outWidthStride *= sizeof(float);
  RetCode code = crop(inData, inHeight, inWidth, 1, inWidthStride, outData,
                      outHeight, outWidth, outWidthStride, left, top, scale,
                      stream);

  return code;
}

template <>
RetCode Crop<float, 3>(hipStream_t stream,
                       int inHeight,
                       int inWidth,
                       int inWidthStride,
                       const float* inData,
                       int outHeight,
                       int outWidth,
                       int outWidthStride,
                       float* outData,
                       const int left,
                       const int top,
                       const float scale) {
  inWidthStride  *= sizeof(float);
  outWidthStride *= sizeof(float);
  RetCode code = crop(inData, inHeight, inWidth, 3, inWidthStride, outData,
                      outHeight, outWidth, outWidthStride, left, top, scale,
                      stream);

  return code;
}

template <>
RetCode Crop<float, 4>(hipStream_t stream,
                       int inHeight,
                       int inWidth,
                       int inWidthStride,
                       const float* inData,
                       int outHeight,
                       int outWidth,
                       int outWidthStride,
                       float* outData,
                       const int left,
                       const int top,
                       const float scale) {
  inWidthStride  *= sizeof(float);
  outWidthStride *= sizeof(float);
  RetCode code = crop(inData, inHeight, inWidth, 4, inWidthStride, outData,
                      outHeight, outWidth, outWidthStride, left, top, scale,
                      stream);

  return code;
}

}  // namespace cuda
}  // namespace cv
}  // namespace ppl
