#include "hip/hip_runtime.h"
/**
 * Licensed to the Apache Software Foundation (ASF) under one or more
 * contributor license agreements. See the NOTICE file distributed with this
 * work for additional information regarding copyright ownership. The ASF
 * licenses this file to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance with the
 * License. You may obtain a copy of the License at
 * http://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 */

#include "ppl/cv/cuda/medianblur.h"

#include <cfloat>

#include "utility.hpp"

using namespace ppl::common;

namespace ppl {
namespace cv {
namespace cuda {

#define RADIUS0 16
#define SMALL_KSIZE0 RADIUS0 * 2 + 1

#define RADIUS1 8
#define SMALL_KSIZE1 RADIUS1 * 2 + 1

template <typename BorderInterpolation>
__global__
void medianC1SharedKernel(const uchar* src, int rows, int cols, int src_stride,
                          int median_index, int radius, uchar* dst,
                          int dst_stride, BorderInterpolation interpolation) {
  __shared__ uchar data[kDimY0 + RADIUS0 * 2][(kDimX0 << 2) + RADIUS0 * 2];

  int element_x = ((blockIdx.x << kShiftX0) + threadIdx.x) << 2;
  int element_y = (blockIdx.y << kShiftY0) + threadIdx.y;

  {
    int index, y_index, row_index;
    uchar* input;
    uchar value0, value1, value2, value3;

    y_index   = threadIdx.y;
    row_index = element_y - radius;
    while (row_index < (int)(((blockIdx.y + 1) << kShiftY0) + radius) &&
           row_index < rows + radius) {
      index = interpolation(rows, radius, row_index);
      input = (uchar*)((uchar*)src + index * src_stride);

      if (threadIdx.x < radius) {
        if (blockIdx.x == 0) {
          index = interpolation(cols, radius, threadIdx.x - radius);
        }
        else {
          index = (blockIdx.x << (kShiftX0 + 2)) + threadIdx.x - radius;
        }
        value0 = input[index];
        data[y_index][threadIdx.x] = value0;
      }

      if (element_x < cols) {
        value0 = input[element_x];
        value1 = input[element_x + 1];
        value2 = input[element_x + 2];
        value3 = input[element_x + 3];
        index = radius + (threadIdx.x << 2);
        data[y_index][index] = value0;
        data[y_index][index + 1] = value1;
        data[y_index][index + 2] = value2;
        data[y_index][index + 3] = value3;
      }

      if (threadIdx.x < radius) {
        index = (cols - radius) >> (kShiftX0 + 2);
        if (blockIdx.x >= index) {
          if (blockIdx.x != gridDim.x - 1) {
            index = ((blockIdx.x + 1) << (kShiftX0 + 2)) + threadIdx.x;
            index = interpolation(cols, radius, index);
            value0 = input[index];
            index = radius + (kDimX0 << 2) + threadIdx.x;
            data[y_index][index] = value0;
          }
          else {
            index = interpolation(cols, radius, cols + threadIdx.x);
            value0 = input[index];
            index = cols - (blockIdx.x << (kShiftX0 + 2));
            index += (radius + threadIdx.x);
            data[y_index][index] = value0;
          }
        }
        else {
          index = ((blockIdx.x + 1) << (kShiftX0 + 2)) + threadIdx.x;
          value0 = input[index];
          index = radius + (kDimX0 << 2) + threadIdx.x;
          data[y_index][index] = value0;
        }
      }

      y_index   += kDimY0;
      row_index += kDimY0;
    }
  }
  __syncthreads();

  if (element_x >= cols || element_y >= rows) {
    return;
  }

  int ksize = (radius << 1) + 1;
  int threadIdx_x = threadIdx.x << 2;
  bool unchecked0, unchecked1, unchecked2, unchecked3;
  uint4 local_count  = make_uint4(0, 0, 0, 0);
  uint4 global_count = make_uint4(0, 0, 0, 0);
  uchar4 value;
  short4 max;
  short4 top = make_short4(256, 256, 256, 256);

  for (int index = 0; index <= median_index; index++) {
    max = make_short4(-1, -1, -1, -1);
    unchecked0 = true;
    unchecked1 = true;
    unchecked2 = true;
    unchecked3 = true;
    for (int i = 0; i < ksize; i++) {
      for (int j = 0; j < ksize; j++) {
        value.x = data[threadIdx.y + i][threadIdx_x + j];
        value.y = data[threadIdx.y + i][threadIdx_x + j + 1];
        value.z = data[threadIdx.y + i][threadIdx_x + j + 2];
        value.w = data[threadIdx.y + i][threadIdx_x + j + 3];
        if ((!unchecked0) && max.x == value.x) unchecked0 = false;
        if ((!unchecked1) && max.y == value.y) unchecked1 = false;
        if ((!unchecked2) && max.z == value.z) unchecked2 = false;
        if ((!unchecked3) && max.w == value.w) unchecked3 = false;

        if (unchecked0 && max.x == value.x) local_count.x++;
        if (unchecked1 && max.y == value.y) local_count.y++;
        if (unchecked2 && max.z == value.z) local_count.z++;
        if (unchecked3 && max.w == value.w) local_count.w++;

        if (index + global_count.x <= median_index && max.x < value.x &&
            value.x < top.x) {
          max.x = value.x;
          local_count.x = 0;
        }
        if (index + global_count.y <= median_index && max.y < value.y &&
            value.y < top.y) {
          max.y = value.y;
          local_count.y = 0;
        }
        if (index + global_count.z <= median_index && max.z < value.z &&
            value.z < top.z) {
          max.z = value.z;
          local_count.z = 0;
        }
        if (index + global_count.w <= median_index && max.w < value.w &&
            value.w < top.w) {
          max.w = value.w;
          local_count.w = 0;
        }
      }
    }
    global_count.x += local_count.x;
    global_count.y += local_count.y;
    global_count.z += local_count.z;
    global_count.w += local_count.w;
    if (max.x != -1) top.x = max.x;
    if (max.y != -1) top.y = max.y;
    if (max.z != -1) top.z = max.z;
    if (max.w != -1) top.w = max.w;
  }

  uchar* output = (uchar*)((uchar*)dst + element_y * dst_stride);
  if (element_x < cols - 3) {
    output[element_x]     = saturateCast(top.x);
    output[element_x + 1] = saturateCast(top.y);
    output[element_x + 2] = saturateCast(top.z);
    output[element_x + 3] = saturateCast(top.w);
  }
  else {
    output[element_x] = saturateCast(top.x);
    if (element_x < cols - 1) {
      output[element_x + 1] = saturateCast(top.y);
    }
    if (element_x < cols - 2) {
      output[element_x + 2] = saturateCast(top.z);
    }
  }
}

template <typename BorderInterpolation>
__global__
void medianC1SharedKernel(const float* src, int rows, int cols, int src_stride,
                          int median_index, int radius, float* dst,
                          int dst_stride, BorderInterpolation interpolation) {
  __shared__ float data[kDimY0 + RADIUS0 * 2][(kDimX0 << 2) + RADIUS0 * 2];

  int element_x = ((blockIdx.x << kShiftX0) + threadIdx.x) << 2;
  int element_y = (blockIdx.y << kShiftY0) + threadIdx.y;

  {
    int index, y_index, row_index;
    float* input;
    float value0, value1, value2, value3;

    y_index   = threadIdx.y;
    row_index = element_y - radius;
    while (row_index < (int)(((blockIdx.y + 1) << kShiftY0) + radius) &&
           row_index < rows + radius) {
      index = interpolation(rows, radius, row_index);
      input = (float*)((uchar*)src + index * src_stride);

      if (threadIdx.x < radius) {
        if (blockIdx.x == 0) {
          index = interpolation(cols, radius, threadIdx.x - radius);
        }
        else {
          index = (blockIdx.x << (kShiftX0 + 2)) + threadIdx.x - radius;
        }
        value0 = input[index];
        data[y_index][threadIdx.x] = value0;
      }

      if (element_x < cols) {
        value0 = input[element_x];
        value1 = input[element_x + 1];
        value2 = input[element_x + 2];
        value3 = input[element_x + 3];
        index = radius + (threadIdx.x << 2);
        data[y_index][index] = value0;
        data[y_index][index + 1] = value1;
        data[y_index][index + 2] = value2;
        data[y_index][index + 3] = value3;
      }

      if (threadIdx.x < radius) {
        index = (cols - radius) >> (kShiftX0 + 2);
        if (blockIdx.x >= index) {
          if (blockIdx.x != gridDim.x - 1) {
            index = ((blockIdx.x + 1) << (kShiftX0 + 2)) + threadIdx.x;
            index = interpolation(cols, radius, index);
            value0 = input[index];
            index = radius + (kDimX0 << 2) + threadIdx.x;
            data[y_index][index] = value0;
          }
          else {
            index = interpolation(cols, radius, cols + threadIdx.x);
            value0 = input[index];
            index = cols - (blockIdx.x << (kShiftX0 + 2));
            index += (radius + threadIdx.x);
            data[y_index][index] = value0;
          }
        }
        else {
          index = ((blockIdx.x + 1) << (kShiftX0 + 2)) + threadIdx.x;
          value0 = input[index];
          index = radius + (kDimX0 << 2) + threadIdx.x;
          data[y_index][index] = value0;
        }
      }

      y_index   += kDimY0;
      row_index += kDimY0;
    }
  }
  __syncthreads();

  if (element_x >= cols || element_y >= rows) {
    return;
  }

  int ksize = (radius << 1) + 1;
  int threadIdx_x = threadIdx.x << 2;
  bool unchecked0, unchecked1, unchecked2, unchecked3;
  uint4 local_count  = make_uint4(0, 0, 0, 0);
  uint4 global_count = make_uint4(0, 0, 0, 0);
  float4 value;
  float4 max;
  float4 top = make_float4(FLT_MAX, FLT_MAX, FLT_MAX, FLT_MAX);

  for (int index = 0; index <= median_index; index++) {
    max = make_float4(FLT_MIN, FLT_MIN, FLT_MIN, FLT_MIN);
    unchecked0 = true;
    unchecked1 = true;
    unchecked2 = true;
    unchecked3 = true;
    for (int i = 0; i < ksize; i++) {
      for (int j = 0; j < ksize; j++) {
        value.x = data[threadIdx.y + i][threadIdx_x + j];
        value.y = data[threadIdx.y + i][threadIdx_x + j + 1];
        value.z = data[threadIdx.y + i][threadIdx_x + j + 2];
        value.w = data[threadIdx.y + i][threadIdx_x + j + 3];
        if ((!unchecked0) && max.x == value.x) unchecked0 = false;
        if ((!unchecked1) && max.y == value.y) unchecked1 = false;
        if ((!unchecked2) && max.z == value.z) unchecked2 = false;
        if ((!unchecked3) && max.w == value.w) unchecked3 = false;

        if (unchecked0 && max.x == value.x) local_count.x++;
        if (unchecked1 && max.y == value.y) local_count.y++;
        if (unchecked2 && max.z == value.z) local_count.z++;
        if (unchecked3 && max.w == value.w) local_count.w++;

        if (index + global_count.x <= median_index && max.x < value.x &&
            value.x < top.x) {
          max.x = value.x;
          local_count.x = 0;
        }
        if (index + global_count.y <= median_index && max.y < value.y &&
            value.y < top.y) {
          max.y = value.y;
          local_count.y = 0;
        }
        if (index + global_count.z <= median_index && max.z < value.z &&
            value.z < top.z) {
          max.z = value.z;
          local_count.z = 0;
        }
        if (index + global_count.w <= median_index && max.w < value.w &&
            value.w < top.w) {
          max.w = value.w;
          local_count.w = 0;
        }
      }
    }
    global_count.x += local_count.x;
    global_count.y += local_count.y;
    global_count.z += local_count.z;
    global_count.w += local_count.w;
    if (max.x != FLT_MIN) top.x = max.x;
    if (max.y != FLT_MIN) top.y = max.y;
    if (max.z != FLT_MIN) top.z = max.z;
    if (max.w != FLT_MIN) top.w = max.w;
  }

  float* output = (float*)((uchar*)dst + element_y * dst_stride);
  if (element_x < cols - 3) {
    output[element_x]     = top.x;
    output[element_x + 1] = top.y;
    output[element_x + 2] = top.z;
    output[element_x + 3] = top.w;
  }
  else {
    output[element_x] = top.x;
    if (element_x < cols - 1) {
      output[element_x + 1] = top.y;
    }
    if (element_x < cols - 2) {
      output[element_x + 2] = top.z;
    }
  }
}

template <typename BorderInterpolation>
__global__
void medianC3SharedKernel(const uchar* src, int rows, int cols, int src_stride,
                          int median_index, int radius, uchar* dst,
                          int dst_stride, BorderInterpolation interpolation) {
  __shared__ uchar3 data[kDimY0 + RADIUS1 * 2][kDimX0 + RADIUS1 * 2];

  int element_x = (blockIdx.x << kShiftX0) + threadIdx.x;
  int element_y = (blockIdx.y << kShiftY0) + threadIdx.y;

  {
    int index, y_index, row_index, col_index;
    uchar3* input;

    y_index   = threadIdx.y;
    row_index = element_y - radius;
    while (row_index < (int)(((blockIdx.y + 1) << kShiftY0) + radius) &&
           row_index < rows + radius) {
      index = interpolation(rows, radius, row_index);
      input = (uchar3*)((uchar*)src + index * src_stride);

      int x_index = threadIdx.x;
      col_index = element_x - radius;
      while (col_index < (int)(((blockIdx.x + 1) << kShiftX0) + radius) &&
            col_index < cols + radius) {
        index = interpolation(cols, radius, col_index);
        data[y_index][x_index] = input[index];
        x_index   += kDimX0;
        col_index += kDimX0;
      }

      y_index   += kDimY0;
      row_index += kDimY0;
    }
  }
  __syncthreads();

  if (element_x >= cols || element_y >= rows) {
    return;
  }

  int ksize = (radius << 1) + 1;
  bool unchecked0, unchecked1, unchecked2;
  uint3 local_count  = make_uint3(0, 0, 0);
  uint3 global_count = make_uint3(0, 0, 0);
  uchar3 value;
  short3 max;
  short3 top = make_short3(256, 256, 256);

  for (int index = 0; index <= median_index; index++) {
    max = make_short3(-1, -1, -1);
    unchecked0 = true;
    unchecked1 = true;
    unchecked2 = true;
    for (int i = 0; i < ksize; i++) {
      for (int j = 0; j < ksize; j++) {
        value = data[threadIdx.y + i][threadIdx.x + j];
        if ((!unchecked0) && max.x == value.x) unchecked0 = false;
        if ((!unchecked1) && max.y == value.y) unchecked1 = false;
        if ((!unchecked2) && max.z == value.z) unchecked2 = false;

        if (unchecked0 && max.x == value.x) local_count.x++;
        if (unchecked1 && max.y == value.y) local_count.y++;
        if (unchecked2 && max.z == value.z) local_count.z++;

        if (index + global_count.x <= median_index && max.x < value.x &&
            value.x < top.x) {
          max.x = value.x;
          local_count.x = 0;
        }
        if (index + global_count.y <= median_index && max.y < value.y &&
            value.y < top.y) {
          max.y = value.y;
          local_count.y = 0;
        }
        if (index + global_count.z <= median_index && max.z < value.z &&
            value.z < top.z) {
          max.z = value.z;
          local_count.z = 0;
        }
      }
    }
    global_count.x += local_count.x;
    global_count.y += local_count.y;
    global_count.z += local_count.z;
    if (max.x != -1) top.x = max.x;
    if (max.y != -1) top.y = max.y;
    if (max.z != -1) top.z = max.z;
  }

  uchar3* output = (uchar3*)((uchar*)dst + element_y * dst_stride);
  output[element_x] = saturateCastVector<uchar3, short3>(top);
}

template <typename BorderInterpolation>
__global__
void medianC3SharedKernel(const float* src, int rows, int cols, int src_stride,
                          int median_index, int radius, float* dst,
                          int dst_stride, BorderInterpolation interpolation) {
  __shared__ float3 data[kDimY0 + RADIUS1 * 2][kDimX0 + RADIUS1 * 2];

  int element_x = (blockIdx.x << kShiftX0) + threadIdx.x;
  int element_y = (blockIdx.y << kShiftY0) + threadIdx.y;

  {
    int index, y_index, row_index, col_index;
    float3* input;

    y_index   = threadIdx.y;
    row_index = element_y - radius;
    while (row_index < (int)(((blockIdx.y + 1) << kShiftY0) + radius) &&
           row_index < rows + radius) {
      index = interpolation(rows, radius, row_index);
      input = (float3*)((uchar*)src + index * src_stride);

      int x_index = threadIdx.x;
      col_index = element_x - radius;
      while (col_index < (int)(((blockIdx.x + 1) << kShiftX0) + radius) &&
             col_index < cols + radius) {
        index = interpolation(cols, radius, col_index);
        data[y_index][x_index] = input[index];
        x_index   += kDimX0;
        col_index += kDimX0;
      }

      y_index   += kDimY0;
      row_index += kDimY0;
    }
  }
  __syncthreads();

  if (element_x >= cols || element_y >= rows) {
    return;
  }

  int ksize = (radius << 1) + 1;
  bool unchecked0, unchecked1, unchecked2;
  uint3 local_count  = make_uint3(0, 0, 0);
  uint3 global_count = make_uint3(0, 0, 0);
  float3 value;
  float3 max;
  float3 top = make_float3(FLT_MAX, FLT_MAX, FLT_MAX);

  for (int index = 0; index <= median_index; index++) {
    max = make_float3(FLT_MIN, FLT_MIN, FLT_MIN);
    unchecked0 = true;
    unchecked1 = true;
    unchecked2 = true;
    for (int i = 0; i < ksize; i++) {
      for (int j = 0; j < ksize; j++) {
        value = data[threadIdx.y + i][threadIdx.x + j];
        if ((!unchecked0) && max.x == value.x) unchecked0 = false;
        if ((!unchecked1) && max.y == value.y) unchecked1 = false;
        if ((!unchecked2) && max.z == value.z) unchecked2 = false;

        if (unchecked0 && max.x == value.x) local_count.x++;
        if (unchecked1 && max.y == value.y) local_count.y++;
        if (unchecked2 && max.z == value.z) local_count.z++;

        if (index + global_count.x <= median_index && max.x < value.x &&
            value.x < top.x) {
          max.x = value.x;
          local_count.x = 0;
        }
        if (index + global_count.y <= median_index && max.y < value.y &&
            value.y < top.y) {
          max.y = value.y;
          local_count.y = 0;
        }
        if (index + global_count.z <= median_index && max.z < value.z &&
            value.z < top.z) {
          max.z = value.z;
          local_count.z = 0;
        }
      }
    }
    global_count.x += local_count.x;
    global_count.y += local_count.y;
    global_count.z += local_count.z;
    if (max.x != FLT_MIN) top.x = max.x;
    if (max.y != FLT_MIN) top.y = max.y;
    if (max.z != FLT_MIN) top.z = max.z;
  }

  float3* output = (float3*)((uchar*)dst + element_y * dst_stride);
  output[element_x] = top;
}

template <typename BorderInterpolation>
__global__
void medianC4SharedKernel(const uchar* src, int rows, int cols, int src_stride,
                          int median_index, int radius, uchar* dst,
                          int dst_stride, BorderInterpolation interpolation) {
  __shared__ uchar4 data[kDimY0 + RADIUS1 * 2][kDimX0 + RADIUS1 * 2];

  int element_x = (blockIdx.x << kShiftX0) + threadIdx.x;
  int element_y = (blockIdx.y << kShiftY0) + threadIdx.y;

  {
    int index, y_index, row_index, col_index;
    uchar4* input;

    y_index   = threadIdx.y;
    row_index = element_y - radius;
    while (row_index < (int)(((blockIdx.y + 1) << kShiftY0) + radius) &&
          row_index < rows + radius) {
      index = interpolation(rows, radius, row_index);
      input = (uchar4*)((uchar*)src + index * src_stride);

      int x_index = threadIdx.x;
      col_index = element_x - radius;
      while (col_index < (int)(((blockIdx.x + 1) << kShiftX0) + radius) &&
            col_index < cols + radius) {
        index = interpolation(cols, radius, col_index);
        data[y_index][x_index] = input[index];
        x_index   += kDimX0;
        col_index += kDimX0;
      }

      y_index   += kDimY0;
      row_index += kDimY0;
    }
  }
  __syncthreads();

  if (element_x >= cols || element_y >= rows) {
    return;
  }

  int ksize = (radius << 1) + 1;
  bool unchecked0, unchecked1, unchecked2, unchecked3;
  uint4 local_count  = make_uint4(0, 0, 0, 0);
  uint4 global_count = make_uint4(0, 0, 0, 0);
  uchar4 value;
  short4 max;
  short4 top = make_short4(256, 256, 256, 256);

  for (int index = 0; index <= median_index; index++) {
    max = make_short4(-1, -1, -1, -1);
    unchecked0 = true;
    unchecked1 = true;
    unchecked2 = true;
    unchecked3 = true;
    for (int i = 0; i < ksize; i++) {
      for (int j = 0; j < ksize; j++) {
        value = data[threadIdx.y + i][threadIdx.x + j];
        if ((!unchecked0) && max.x == value.x) unchecked0 = false;
        if ((!unchecked1) && max.y == value.y) unchecked1 = false;
        if ((!unchecked2) && max.z == value.z) unchecked2 = false;
        if ((!unchecked3) && max.w == value.w) unchecked3 = false;

        if (unchecked0 && max.x == value.x) local_count.x++;
        if (unchecked1 && max.y == value.y) local_count.y++;
        if (unchecked2 && max.z == value.z) local_count.z++;
        if (unchecked3 && max.w == value.w) local_count.w++;

        if (index + global_count.x <= median_index && max.x < value.x &&
            value.x < top.x) {
          max.x = value.x;
          local_count.x = 0;
        }
        if (index + global_count.y <= median_index && max.y < value.y &&
            value.y < top.y) {
          max.y = value.y;
          local_count.y = 0;
        }
        if (index + global_count.z <= median_index && max.z < value.z &&
            value.z < top.z) {
          max.z = value.z;
          local_count.z = 0;
        }
        if (index + global_count.w <= median_index && max.w < value.w &&
            value.w < top.w) {
          max.w = value.w;
          local_count.w = 0;
        }
      }
    }
    global_count.x += local_count.x;
    global_count.y += local_count.y;
    global_count.z += local_count.z;
    global_count.w += local_count.w;
    if (max.x != -1) top.x = max.x;
    if (max.y != -1) top.y = max.y;
    if (max.z != -1) top.z = max.z;
    if (max.w != -1) top.w = max.w;
  }

  uchar4* output = (uchar4*)((uchar*)dst + element_y * dst_stride);
  output[element_x] = saturateCastVector<uchar4, short4>(top);
}

template <typename BorderInterpolation>
__global__
void medianC4SharedKernel(const float* src, int rows, int cols, int src_stride,
                          int median_index, int radius, float* dst,
                          int dst_stride, BorderInterpolation interpolation) {
  __shared__ float4 data[kDimY0 + RADIUS1 * 2][kDimX0 + RADIUS1 * 2];

  int element_x = (blockIdx.x << kShiftX0) + threadIdx.x;
  int element_y = (blockIdx.y << kShiftY0) + threadIdx.y;

  {
    int index, y_index, row_index, col_index;
    float4* input;

    y_index   = threadIdx.y;
    row_index = element_y - radius;
    while (row_index < (int)(((blockIdx.y + 1) << kShiftY0) + radius) &&
          row_index < rows + radius) {
      index = interpolation(rows, radius, row_index);
      input = (float4*)((uchar*)src + index * src_stride);

      int x_index = threadIdx.x;
      col_index = element_x - radius;
      while (col_index < (int)(((blockIdx.x + 1) << kShiftX0) + radius) &&
             col_index < cols + radius) {
        index = interpolation(cols, radius, col_index);
        data[y_index][x_index] = input[index];
        x_index   += kDimX0;
        col_index += kDimX0;
      }

      y_index   += kDimY0;
      row_index += kDimY0;
    }
  }
  __syncthreads();

  if (element_x >= cols || element_y >= rows) {
    return;
  }

  int ksize = (radius << 1) + 1;
  bool unchecked0, unchecked1, unchecked2, unchecked3;
  uint4 local_count  = make_uint4(0, 0, 0, 0);
  uint4 global_count = make_uint4(0, 0, 0, 0);
  float4 value;
  float4 max;
  float4 top = make_float4(FLT_MAX, FLT_MAX, FLT_MAX, FLT_MAX);

  for (int index = 0; index <= median_index; index++) {
    max = make_float4(FLT_MIN, FLT_MIN, FLT_MIN, FLT_MIN);
    unchecked0 = true;
    unchecked1 = true;
    unchecked2 = true;
    unchecked3 = true;
    for (int i = 0; i < ksize; i++) {
      for (int j = 0; j < ksize; j++) {
        value = data[threadIdx.y + i][threadIdx.x + j];
        if ((!unchecked0) && max.x == value.x) unchecked0 = false;
        if ((!unchecked1) && max.y == value.y) unchecked1 = false;
        if ((!unchecked2) && max.z == value.z) unchecked2 = false;
        if ((!unchecked3) && max.w == value.w) unchecked3 = false;

        if (unchecked0 && max.x == value.x) local_count.x++;
        if (unchecked1 && max.y == value.y) local_count.y++;
        if (unchecked2 && max.z == value.z) local_count.z++;
        if (unchecked3 && max.w == value.w) local_count.w++;

        if (index + global_count.x <= median_index && max.x < value.x &&
            value.x < top.x) {
          max.x = value.x;
          local_count.x = 0;
        }
        if (index + global_count.y <= median_index && max.y < value.y &&
            value.y < top.y) {
          max.y = value.y;
          local_count.y = 0;
        }
        if (index + global_count.z <= median_index && max.z < value.z &&
            value.z < top.z) {
          max.z = value.z;
          local_count.z = 0;
        }
        if (index + global_count.w <= median_index && max.w < value.w &&
            value.w < top.w) {
          max.w = value.w;
          local_count.w = 0;
        }
      }
    }
    global_count.x += local_count.x;
    global_count.y += local_count.y;
    global_count.z += local_count.z;
    global_count.w += local_count.w;
    if (max.x != FLT_MIN) top.x = max.x;
    if (max.y != FLT_MIN) top.y = max.y;
    if (max.z != FLT_MIN) top.z = max.z;
    if (max.w != FLT_MIN) top.w = max.w;
  }

  float4* output = (float4*)((uchar*)dst + element_y * dst_stride);
  output[element_x] = top;
}

template <typename BorderInterpolation>
__global__
void medianC1Kernel(const uchar* src, int rows, int cols, int src_stride,
                    int median_index, int radius, uchar* dst, int dst_stride,
                    BorderInterpolation interpolation) {
  int element_x = ((blockIdx.x << kBlockShiftX0) + threadIdx.x) << 2;
  int element_y = (blockIdx.y << kBlockShiftY0) + threadIdx.y;
  if (element_x >= cols || element_y >= rows) {
    return;
  }

  int origin_x = element_x - radius;
  int origin_y = element_y - radius;
  int top_x    = element_x + radius;
  int top_y    = element_y + radius;

  int data_index;
  bool unchecked0, unchecked1, unchecked2, unchecked3;
  uint4 local_count  = make_uint4(0, 0, 0, 0);
  uint4 global_count = make_uint4(0, 0, 0, 0);
  uchar* input;
  uchar4 value;
  short4 max;
  short4 top = make_short4(256, 256, 256, 256);

  bool isnt_border_block = true;
  data_index = radius >> (kBlockShiftX0 + 2);
  if (blockIdx.x <= data_index) isnt_border_block = false;
  data_index = (cols - radius) >> (kBlockShiftX0 + 2);
  if (blockIdx.x >= data_index) isnt_border_block = false;

  if (isnt_border_block) {
    for (int index = 0; index <= median_index; index++) {
      max = make_short4(-1, -1, -1, -1);
      unchecked0 = true;
      unchecked1 = true;
      unchecked2 = true;
      unchecked3 = true;
      for (int i = origin_y; i <= top_y; i++) {
        data_index = interpolation(rows, radius, i);
        input = (uchar*)((uchar*)src + data_index * src_stride);
        for (int j = origin_x; j <= top_x; j++) {
          value.x = input[j];
          value.y = input[j + 1];
          value.z = input[j + 2];
          value.w = input[j + 3];
          if ((!unchecked0) && max.x == value.x) unchecked0 = false;
          if ((!unchecked1) && max.y == value.y) unchecked1 = false;
          if ((!unchecked2) && max.z == value.z) unchecked2 = false;
          if ((!unchecked3) && max.w == value.w) unchecked3 = false;

          if (unchecked0 && max.x == value.x) local_count.x++;
          if (unchecked1 && max.y == value.y) local_count.y++;
          if (unchecked2 && max.z == value.z) local_count.z++;
          if (unchecked3 && max.w == value.w) local_count.w++;

          if (index + global_count.x <= median_index && max.x < value.x &&
              value.x < top.x) {
            max.x = value.x;
            local_count.x = 0;
          }
          if (index + global_count.y <= median_index && max.y < value.y &&
              value.y < top.y) {
            max.y = value.y;
            local_count.y = 0;
          }
          if (index + global_count.z <= median_index && max.z < value.z &&
              value.z < top.z) {
            max.z = value.z;
            local_count.z = 0;
          }
          if (index + global_count.w <= median_index && max.w < value.w &&
              value.w < top.w) {
            max.w = value.w;
            local_count.w = 0;
          }
        }
      }
      global_count.x += local_count.x;
      global_count.y += local_count.y;
      global_count.z += local_count.z;
      global_count.w += local_count.w;
      if (max.x != -1) top.x = max.x;
      if (max.y != -1) top.y = max.y;
      if (max.z != -1) top.z = max.z;
      if (max.w != -1) top.w = max.w;
    }
  }
  else {
    for (int index = 0; index <= median_index; index++) {
      max = make_short4(-1, -1, -1, -1);
      unchecked0 = true;
      unchecked1 = true;
      unchecked2 = true;
      unchecked3 = true;
      for (int i = origin_y; i <= top_y; i++) {
        data_index = interpolation(rows, radius, i);
        input = (uchar*)((uchar*)src + data_index * src_stride);
        for (int j = origin_x; j <= top_x; j++) {
          data_index = interpolation(cols, radius, j);
          value.x = input[data_index];
          data_index = interpolation(cols, radius, j + 1);
          value.y = input[data_index];
          data_index = interpolation(cols, radius, j + 2);
          value.z = input[data_index];
          data_index = interpolation(cols, radius, j + 3);
          value.w = input[data_index];
          if ((!unchecked0) && max.x == value.x) unchecked0 = false;
          if ((!unchecked1) && max.y == value.y) unchecked1 = false;
          if ((!unchecked2) && max.z == value.z) unchecked2 = false;
          if ((!unchecked3) && max.w == value.w) unchecked3 = false;

          if (unchecked0 && max.x == value.x) local_count.x++;
          if (unchecked1 && max.y == value.y) local_count.y++;
          if (unchecked2 && max.z == value.z) local_count.z++;
          if (unchecked3 && max.w == value.w) local_count.w++;

          if (index + global_count.x <= median_index && max.x < value.x &&
              value.x < top.x) {
            max.x = value.x;
            local_count.x = 0;
          }
          if (index + global_count.y <= median_index && max.y < value.y &&
              value.y < top.y) {
            max.y = value.y;
            local_count.y = 0;
          }
          if (index + global_count.z <= median_index && max.z < value.z &&
              value.z < top.z) {
            max.z = value.z;
            local_count.z = 0;
          }
          if (index + global_count.w <= median_index && max.w < value.w &&
              value.w < top.w) {
            max.w = value.w;
            local_count.w = 0;
          }
        }
      }
      global_count.x += local_count.x;
      global_count.y += local_count.y;
      global_count.z += local_count.z;
      global_count.w += local_count.w;
      if (max.x != -1) top.x = max.x;
      if (max.y != -1) top.y = max.y;
      if (max.z != -1) top.z = max.z;
      if (max.w != -1) top.w = max.w;
    }
  }

  uchar* output = (uchar*)((uchar*)dst + element_y * dst_stride);
  if (element_x < cols - 3) {
    output[element_x]     = saturateCast(top.x);
    output[element_x + 1] = saturateCast(top.y);
    output[element_x + 2] = saturateCast(top.z);
    output[element_x + 3] = saturateCast(top.w);
  }
  else {
    output[element_x] = saturateCast(top.x);
    if (element_x < cols - 1) {
      output[element_x + 1] = saturateCast(top.y);
    }
    if (element_x < cols - 2) {
      output[element_x + 2] = saturateCast(top.z);
    }
  }
}

template <typename BorderInterpolation>
__global__
void medianC1Kernel(const float* src, int rows, int cols, int src_stride,
                    int median_index, int radius, float* dst, int dst_stride,
                    BorderInterpolation interpolation) {
  int element_x = ((blockIdx.x << kBlockShiftX1) + threadIdx.x) << 2;
  int element_y = (blockIdx.y << kBlockShiftY1) + threadIdx.y;
  if (element_x >= cols || element_y >= rows) {
    return;
  }

  int origin_x = element_x - radius;
  int origin_y = element_y - radius;
  int top_x    = element_x + radius;
  int top_y    = element_y + radius;

  int data_index;
  bool unchecked0, unchecked1, unchecked2, unchecked3;
  uint4 local_count  = make_uint4(0, 0, 0, 0);
  uint4 global_count = make_uint4(0, 0, 0, 0);
  float* input;
  float4 value;
  float4 max;
  float4 top = make_float4(FLT_MAX, FLT_MAX, FLT_MAX, FLT_MAX);

  bool isnt_border_block = true;
  data_index = radius >> (kBlockShiftX0 + 2);
  if (blockIdx.x <= data_index) isnt_border_block = false;
  data_index = (cols - radius) >> (kBlockShiftX0 + 2);
  if (blockIdx.x >= data_index) isnt_border_block = false;

  if (isnt_border_block) {
    for (int index = 0; index <= median_index; index++) {
      max = make_float4(FLT_MIN, FLT_MIN, FLT_MIN, FLT_MIN);
      unchecked0 = true;
      unchecked1 = true;
      unchecked2 = true;
      unchecked3 = true;
      for (int i = origin_y; i <= top_y; i++) {
        data_index = interpolation(rows, radius, i);
        input = (float*)((uchar*)src + data_index * src_stride);
        for (int j = origin_x; j <= top_x; j++) {
          value.x = input[j];
          value.y = input[j + 1];
          value.z = input[j + 2];
          value.w = input[j + 3];
          if ((!unchecked0) && max.x == value.x) unchecked0 = false;
          if ((!unchecked1) && max.y == value.y) unchecked1 = false;
          if ((!unchecked2) && max.z == value.z) unchecked2 = false;
          if ((!unchecked3) && max.w == value.w) unchecked3 = false;

          if (unchecked0 && max.x == value.x) local_count.x++;
          if (unchecked1 && max.y == value.y) local_count.y++;
          if (unchecked2 && max.z == value.z) local_count.z++;
          if (unchecked3 && max.w == value.w) local_count.w++;

          if (index + global_count.x <= median_index && max.x < value.x &&
              value.x < top.x) {
            max.x = value.x;
            local_count.x = 0;
          }
          if (index + global_count.y <= median_index && max.y < value.y &&
              value.y < top.y) {
            max.y = value.y;
            local_count.y = 0;
          }
          if (index + global_count.z <= median_index && max.z < value.z &&
              value.z < top.z) {
            max.z = value.z;
            local_count.z = 0;
          }
          if (index + global_count.w <= median_index && max.w < value.w &&
              value.w < top.w) {
            max.w = value.w;
            local_count.w = 0;
          }
        }
      }
      global_count.x += local_count.x;
      global_count.y += local_count.y;
      global_count.z += local_count.z;
      global_count.w += local_count.w;
      if (max.x != FLT_MIN) top.x = max.x;
      if (max.y != FLT_MIN) top.y = max.y;
      if (max.z != FLT_MIN) top.z = max.z;
      if (max.w != FLT_MIN) top.w = max.w;
    }
  }
  else {
    for (int index = 0; index <= median_index; index++) {
      max = make_float4(FLT_MIN, FLT_MIN, FLT_MIN, FLT_MIN);
      unchecked0 = true;
      unchecked1 = true;
      unchecked2 = true;
      unchecked3 = true;
      for (int i = origin_y; i <= top_y; i++) {
        data_index = interpolation(rows, radius, i);
        input = (float*)((uchar*)src + data_index * src_stride);
        for (int j = origin_x; j <= top_x; j++) {
          data_index = interpolation(cols, radius, j);
          value.x = input[data_index];
          data_index = interpolation(cols, radius, j + 1);
          value.y = input[data_index];
          data_index = interpolation(cols, radius, j + 2);
          value.z = input[data_index];
          data_index = interpolation(cols, radius, j + 3);
          value.w = input[data_index];
          if ((!unchecked0) && max.x == value.x) unchecked0 = false;
          if ((!unchecked1) && max.y == value.y) unchecked1 = false;
          if ((!unchecked2) && max.z == value.z) unchecked2 = false;
          if ((!unchecked3) && max.w == value.w) unchecked3 = false;

          if (unchecked0 && max.x == value.x) local_count.x++;
          if (unchecked1 && max.y == value.y) local_count.y++;
          if (unchecked2 && max.z == value.z) local_count.z++;
          if (unchecked3 && max.w == value.w) local_count.w++;

          if (index + global_count.x <= median_index && max.x < value.x &&
              value.x < top.x) {
            max.x = value.x;
            local_count.x = 0;
          }
          if (index + global_count.y <= median_index && max.y < value.y &&
              value.y < top.y) {
            max.y = value.y;
            local_count.y = 0;
          }
          if (index + global_count.z <= median_index && max.z < value.z &&
              value.z < top.z) {
            max.z = value.z;
            local_count.z = 0;
          }
          if (index + global_count.w <= median_index && max.w < value.w &&
              value.w < top.w) {
            max.w = value.w;
            local_count.w = 0;
          }
        }
      }
      global_count.x += local_count.x;
      global_count.y += local_count.y;
      global_count.z += local_count.z;
      global_count.w += local_count.w;
      if (max.x != FLT_MIN) top.x = max.x;
      if (max.y != FLT_MIN) top.y = max.y;
      if (max.z != FLT_MIN) top.z = max.z;
      if (max.w != FLT_MIN) top.w = max.w;
    }
  }

  float* output = (float*)((uchar*)dst + element_y * dst_stride);
  if (element_x < cols - 3) {
    output[element_x]     = top.x;
    output[element_x + 1] = top.y;
    output[element_x + 2] = top.z;
    output[element_x + 3] = top.w;
  }
  else {
    output[element_x] = top.x;
    if (element_x < cols - 1) {
      output[element_x + 1] = top.y;
    }
    if (element_x < cols - 2) {
      output[element_x + 2] = top.z;
    }
  }
}

template <typename BorderInterpolation>
__global__
void medianC3Kernel(const uchar* src, int rows, int cols, int src_stride,
                    int median_index, int radius, uchar* dst, int dst_stride,
                    BorderInterpolation interpolation) {
  int element_x = (blockIdx.x << kBlockShiftX0) + threadIdx.x;
  int element_y = (blockIdx.y << kBlockShiftY0) + threadIdx.y;
  if (element_x >= cols || element_y >= rows) {
    return;
  }

  int origin_x = element_x - radius;
  int origin_y = element_y - radius;
  int top_x    = element_x + radius;
  int top_y    = element_y + radius;

  int data_index;
  bool unchecked0, unchecked1, unchecked2;
  uint3 local_count  = make_uint3(0, 0, 0);
  uint3 global_count = make_uint3(0, 0, 0);
  uchar3* input;
  uchar3 value;
  short3 max;
  short3 top = make_short3(256, 256, 256);

  bool isnt_border_block = true;
  data_index = radius >> (kBlockShiftX0 + 2);
  if (blockIdx.x <= data_index) isnt_border_block = false;
  data_index = (cols - radius) >> (kBlockShiftX0 + 2);
  if (blockIdx.x >= data_index) isnt_border_block = false;

  if (isnt_border_block) {
    for (int index = 0; index <= median_index; index++) {
      max = make_short3(-1, -1, -1);
      unchecked0 = true;
      unchecked1 = true;
      unchecked2 = true;
      for (int i = origin_y; i <= top_y; i++) {
        data_index = interpolation(rows, radius, i);
        input = (uchar3*)((uchar*)src + data_index * src_stride);
        for (int j = origin_x; j <= top_x; j++) {
          value = input[j];
          if ((!unchecked0) && max.x == value.x) unchecked0 = false;
          if ((!unchecked1) && max.y == value.y) unchecked1 = false;
          if ((!unchecked2) && max.z == value.z) unchecked2 = false;

          if (unchecked0 && max.x == value.x) local_count.x++;
          if (unchecked1 && max.y == value.y) local_count.y++;
          if (unchecked2 && max.z == value.z) local_count.z++;

          if (index + global_count.x <= median_index && max.x < value.x &&
              value.x < top.x) {
            max.x = value.x;
            local_count.x = 0;
          }
          if (index + global_count.y <= median_index && max.y < value.y &&
              value.y < top.y) {
            max.y = value.y;
            local_count.y = 0;
          }
          if (index + global_count.z <= median_index && max.z < value.z &&
              value.z < top.z) {
            max.z = value.z;
            local_count.z = 0;
          }
        }
      }
      global_count.x += local_count.x;
      global_count.y += local_count.y;
      global_count.z += local_count.z;
      if (max.x != -1) top.x = max.x;
      if (max.y != -1) top.y = max.y;
      if (max.z != -1) top.z = max.z;
    }
  }
  else {
    for (int index = 0; index <= median_index; index++) {
      max = make_short3(-1, -1, -1);
      unchecked0 = true;
      unchecked1 = true;
      unchecked2 = true;
      for (int i = origin_y; i <= top_y; i++) {
        data_index = interpolation(rows, radius, i);
        input = (uchar3*)((uchar*)src + data_index * src_stride);
        for (int j = origin_x; j <= top_x; j++) {
          data_index = interpolation(cols, radius, j);
          value = input[data_index];
          if ((!unchecked0) && max.x == value.x) unchecked0 = false;
          if ((!unchecked1) && max.y == value.y) unchecked1 = false;
          if ((!unchecked2) && max.z == value.z) unchecked2 = false;

          if (unchecked0 && max.x == value.x) local_count.x++;
          if (unchecked1 && max.y == value.y) local_count.y++;
          if (unchecked2 && max.z == value.z) local_count.z++;

          if (index + global_count.x <= median_index && max.x < value.x &&
              value.x < top.x) {
            max.x = value.x;
            local_count.x = 0;
          }
          if (index + global_count.y <= median_index && max.y < value.y &&
              value.y < top.y) {
            max.y = value.y;
            local_count.y = 0;
          }
          if (index + global_count.z <= median_index && max.z < value.z &&
              value.z < top.z) {
            max.z = value.z;
            local_count.z = 0;
          }
        }
      }
      global_count.x += local_count.x;
      global_count.y += local_count.y;
      global_count.z += local_count.z;
      if (max.x != -1) top.x = max.x;
      if (max.y != -1) top.y = max.y;
      if (max.z != -1) top.z = max.z;
    }
  }

  uchar3* output = (uchar3*)((uchar*)dst + element_y * dst_stride);
  output[element_x] = saturateCastVector<uchar3, short3>(top);
}

template <typename BorderInterpolation>
__global__
void medianC3Kernel(const float* src, int rows, int cols, int src_stride,
                    int median_index, int radius, float* dst, int dst_stride,
                    BorderInterpolation interpolation) {
  int element_x = (blockIdx.x << kBlockShiftX1) + threadIdx.x;
  int element_y = (blockIdx.y << kBlockShiftY1) + threadIdx.y;
  if (element_x >= cols || element_y >= rows) {
    return;
  }

  int origin_x = element_x - radius;
  int origin_y = element_y - radius;
  int top_x    = element_x + radius;
  int top_y    = element_y + radius;

  int data_index;
  bool unchecked0, unchecked1, unchecked2;
  uint3 local_count  = make_uint3(0, 0, 0);
  uint3 global_count = make_uint3(0, 0, 0);
  float3* input;
  float3 value;
  float3 max;
  float3 top = make_float3(FLT_MAX, FLT_MAX, FLT_MAX);

  bool isnt_border_block = true;
  data_index = radius >> (kBlockShiftX0 + 2);
  if (blockIdx.x <= data_index) isnt_border_block = false;
  data_index = (cols - radius) >> (kBlockShiftX0 + 2);
  if (blockIdx.x >= data_index) isnt_border_block = false;

  if (isnt_border_block) {
    for (int index = 0; index <= median_index; index++) {
      max = make_float3(FLT_MIN, FLT_MIN, FLT_MIN);
      unchecked0 = true;
      unchecked1 = true;
      unchecked2 = true;
      for (int i = origin_y; i <= top_y; i++) {
        data_index = interpolation(rows, radius, i);
        input = (float3*)((uchar*)src + data_index * src_stride);
        for (int j = origin_x; j <= top_x; j++) {
          value = input[j];
          if ((!unchecked0) && max.x == value.x) unchecked0 = false;
          if ((!unchecked1) && max.y == value.y) unchecked1 = false;
          if ((!unchecked2) && max.z == value.z) unchecked2 = false;

          if (unchecked0 && max.x == value.x) local_count.x++;
          if (unchecked1 && max.y == value.y) local_count.y++;
          if (unchecked2 && max.z == value.z) local_count.z++;

          if (index + global_count.x <= median_index && max.x < value.x &&
              value.x < top.x) {
            max.x = value.x;
            local_count.x = 0;
          }
          if (index + global_count.y <= median_index && max.y < value.y &&
              value.y < top.y) {
            max.y = value.y;
            local_count.y = 0;
          }
          if (index + global_count.z <= median_index && max.z < value.z &&
              value.z < top.z) {
            max.z = value.z;
            local_count.z = 0;
          }
        }
      }
      global_count.x += local_count.x;
      global_count.y += local_count.y;
      global_count.z += local_count.z;
      if (max.x != FLT_MIN) top.x = max.x;
      if (max.y != FLT_MIN) top.y = max.y;
      if (max.z != FLT_MIN) top.z = max.z;
    }
  }
  else {
    for (int index = 0; index <= median_index; index++) {
      max = make_float3(FLT_MIN, FLT_MIN, FLT_MIN);
      unchecked0 = true;
      unchecked1 = true;
      unchecked2 = true;
      for (int i = origin_y; i <= top_y; i++) {
        data_index = interpolation(rows, radius, i);
        input = (float3*)((uchar*)src + data_index * src_stride);
        for (int j = origin_x; j <= top_x; j++) {
          data_index = interpolation(cols, radius, j);
          value = input[data_index];
          if ((!unchecked0) && max.x == value.x) unchecked0 = false;
          if ((!unchecked1) && max.y == value.y) unchecked1 = false;
          if ((!unchecked2) && max.z == value.z) unchecked2 = false;

          if (unchecked0 && max.x == value.x) local_count.x++;
          if (unchecked1 && max.y == value.y) local_count.y++;
          if (unchecked2 && max.z == value.z) local_count.z++;

          if (index + global_count.x <= median_index && max.x < value.x &&
              value.x < top.x) {
            max.x = value.x;
            local_count.x = 0;
          }
          if (index + global_count.y <= median_index && max.y < value.y &&
              value.y < top.y) {
            max.y = value.y;
            local_count.y = 0;
          }
          if (index + global_count.z <= median_index && max.z < value.z &&
              value.z < top.z) {
            max.z = value.z;
            local_count.z = 0;
          }
        }
      }
      global_count.x += local_count.x;
      global_count.y += local_count.y;
      global_count.z += local_count.z;
      if (max.x != FLT_MIN) top.x = max.x;
      if (max.y != FLT_MIN) top.y = max.y;
      if (max.z != FLT_MIN) top.z = max.z;
    }
  }

  float3* output = (float3*)((uchar*)dst + element_y * dst_stride);
  output[element_x] = top;
}

template <typename BorderInterpolation>
__global__
void medianC4Kernel(const uchar* src, int rows, int cols, int src_stride,
                    int median_index, int radius, uchar* dst, int dst_stride,
                    BorderInterpolation interpolation) {
  int element_x = (blockIdx.x << kBlockShiftX0) + threadIdx.x;
  int element_y = (blockIdx.y << kBlockShiftY0) + threadIdx.y;
  if (element_x >= cols || element_y >= rows) {
    return;
  }

  int origin_x = element_x - radius;
  int origin_y = element_y - radius;
  int top_x    = element_x + radius;
  int top_y    = element_y + radius;

  int data_index;
  bool unchecked0, unchecked1, unchecked2, unchecked3;
  uint4 local_count  = make_uint4(0, 0, 0, 0);
  uint4 global_count = make_uint4(0, 0, 0, 0);
  uchar4* input;
  uchar4 value;
  short4 max;
  short4 top = make_short4(256, 256, 256, 256);

  bool isnt_border_block = true;
  data_index = radius >> (kBlockShiftX0 + 2);
  if (blockIdx.x <= data_index) isnt_border_block = false;
  data_index = (cols - radius) >> (kBlockShiftX0 + 2);
  if (blockIdx.x >= data_index) isnt_border_block = false;

  if (isnt_border_block) {
    for (int index = 0; index <= median_index; index++) {
      max = make_short4(-1, -1, -1, -1);
      unchecked0 = true;
      unchecked1 = true;
      unchecked2 = true;
      unchecked3 = true;
      for (int i = origin_y; i <= top_y; i++) {
        data_index = interpolation(rows, radius, i);
        input = (uchar4*)((uchar*)src + data_index * src_stride);
        for (int j = origin_x; j <= top_x; j++) {
          value = input[j];
          if ((!unchecked0) && max.x == value.x) unchecked0 = false;
          if ((!unchecked1) && max.y == value.y) unchecked1 = false;
          if ((!unchecked2) && max.z == value.z) unchecked2 = false;
          if ((!unchecked3) && max.w == value.w) unchecked3 = false;

          if (unchecked0 && max.x == value.x) local_count.x++;
          if (unchecked1 && max.y == value.y) local_count.y++;
          if (unchecked2 && max.z == value.z) local_count.z++;
          if (unchecked3 && max.w == value.w) local_count.w++;

          if (index + global_count.x <= median_index && max.x < value.x &&
              value.x < top.x) {
            max.x = value.x;
            local_count.x = 0;
          }
          if (index + global_count.y <= median_index && max.y < value.y &&
              value.y < top.y) {
            max.y = value.y;
            local_count.y = 0;
          }
          if (index + global_count.z <= median_index && max.z < value.z &&
              value.z < top.z) {
            max.z = value.z;
            local_count.z = 0;
          }
          if (index + global_count.w <= median_index && max.w < value.w &&
              value.w < top.w) {
            max.w = value.w;
            local_count.w = 0;
          }
        }
      }
      global_count.x += local_count.x;
      global_count.y += local_count.y;
      global_count.z += local_count.z;
      global_count.w += local_count.w;
      if (max.x != -1) top.x = max.x;
      if (max.y != -1) top.y = max.y;
      if (max.z != -1) top.z = max.z;
      if (max.w != -1) top.w = max.w;
    }
  }
  else {
    for (int index = 0; index <= median_index; index++) {
      max = make_short4(-1, -1, -1, -1);
      unchecked0 = true;
      unchecked1 = true;
      unchecked2 = true;
      unchecked3 = true;
      for (int i = origin_y; i <= top_y; i++) {
        data_index = interpolation(rows, radius, i);
        input = (uchar4*)((uchar*)src + data_index * src_stride);
        for (int j = origin_x; j <= top_x; j++) {
          data_index = interpolation(cols, radius, j);
          value = input[data_index];
          if ((!unchecked0) && max.x == value.x) unchecked0 = false;
          if ((!unchecked1) && max.y == value.y) unchecked1 = false;
          if ((!unchecked2) && max.z == value.z) unchecked2 = false;
          if ((!unchecked3) && max.w == value.w) unchecked3 = false;

          if (unchecked0 && max.x == value.x) local_count.x++;
          if (unchecked1 && max.y == value.y) local_count.y++;
          if (unchecked2 && max.z == value.z) local_count.z++;
          if (unchecked3 && max.w == value.w) local_count.w++;

          if (index + global_count.x <= median_index && max.x < value.x &&
              value.x < top.x) {
            max.x = value.x;
            local_count.x = 0;
          }
          if (index + global_count.y <= median_index && max.y < value.y &&
              value.y < top.y) {
            max.y = value.y;
            local_count.y = 0;
          }
          if (index + global_count.z <= median_index && max.z < value.z &&
              value.z < top.z) {
            max.z = value.z;
            local_count.z = 0;
          }
          if (index + global_count.w <= median_index && max.w < value.w &&
              value.w < top.w) {
            max.w = value.w;
            local_count.w = 0;
          }
        }
      }
      global_count.x += local_count.x;
      global_count.y += local_count.y;
      global_count.z += local_count.z;
      global_count.w += local_count.w;
      if (max.x != -1) top.x = max.x;
      if (max.y != -1) top.y = max.y;
      if (max.z != -1) top.z = max.z;
      if (max.w != -1) top.w = max.w;
    }
  }

  uchar4* output = (uchar4*)((uchar*)dst + element_y * dst_stride);
  output[element_x] = saturateCastVector<uchar4, short4>(top);
}

template <typename BorderInterpolation>
__global__
void medianC4Kernel(const float* src, int rows, int cols, int src_stride,
                    int median_index, int radius, float* dst, int dst_stride,
                    BorderInterpolation interpolation) {
  int element_x = (blockIdx.x << kBlockShiftX1) + threadIdx.x;
  int element_y = (blockIdx.y << kBlockShiftY1) + threadIdx.y;
  if (element_x >= cols || element_y >= rows) {
    return;
  }

  int origin_x = element_x - radius;
  int origin_y = element_y - radius;
  int top_x    = element_x + radius;
  int top_y    = element_y + radius;

  int data_index;
  bool unchecked0, unchecked1, unchecked2, unchecked3;
  uint4 local_count  = make_uint4(0, 0, 0, 0);
  uint4 global_count = make_uint4(0, 0, 0, 0);
  float4* input;
  float4 value;
  float4 max;
  float4 top = make_float4(FLT_MAX, FLT_MAX, FLT_MAX, FLT_MAX);

  bool isnt_border_block = true;
  data_index = radius >> (kBlockShiftX0 + 2);
  if (blockIdx.x <= data_index) isnt_border_block = false;
  data_index = (cols - radius) >> (kBlockShiftX0 + 2);
  if (blockIdx.x >= data_index) isnt_border_block = false;

  if (isnt_border_block) {
    for (int index = 0; index <= median_index; index++) {
      max = make_float4(FLT_MIN, FLT_MIN, FLT_MIN, FLT_MIN);
      unchecked0 = true;
      unchecked1 = true;
      unchecked2 = true;
      unchecked3 = true;
      for (int i = origin_y; i <= top_y; i++) {
        data_index = interpolation(rows, radius, i);
        input = (float4*)((uchar*)src + data_index * src_stride);
        for (int j = origin_x; j <= top_x; j++) {
          value = input[j];
          if ((!unchecked0) && max.x == value.x) unchecked0 = false;
          if ((!unchecked1) && max.y == value.y) unchecked1 = false;
          if ((!unchecked2) && max.z == value.z) unchecked2 = false;
          if ((!unchecked3) && max.w == value.w) unchecked3 = false;

          if (unchecked0 && max.x == value.x) local_count.x++;
          if (unchecked1 && max.y == value.y) local_count.y++;
          if (unchecked2 && max.z == value.z) local_count.z++;
          if (unchecked3 && max.w == value.w) local_count.w++;

          if (index + global_count.x <= median_index && max.x < value.x &&
              value.x < top.x) {
            max.x = value.x;
            local_count.x = 0;
          }
          if (index + global_count.y <= median_index && max.y < value.y &&
              value.y < top.y) {
            max.y = value.y;
            local_count.y = 0;
          }
          if (index + global_count.z <= median_index && max.z < value.z &&
              value.z < top.z) {
            max.z = value.z;
            local_count.z = 0;
          }
          if (index + global_count.w <= median_index && max.w < value.w &&
              value.w < top.w) {
            max.w = value.w;
            local_count.w = 0;
          }
        }
      }
      global_count.x += local_count.x;
      global_count.y += local_count.y;
      global_count.z += local_count.z;
      global_count.w += local_count.w;
      if (max.x != FLT_MIN) top.x = max.x;
      if (max.y != FLT_MIN) top.y = max.y;
      if (max.z != FLT_MIN) top.z = max.z;
      if (max.w != FLT_MIN) top.w = max.w;
    }
  }
  else {
    for (int index = 0; index <= median_index; index++) {
      max = make_float4(FLT_MIN, FLT_MIN, FLT_MIN, FLT_MIN);
      unchecked0 = true;
      unchecked1 = true;
      unchecked2 = true;
      unchecked3 = true;
      for (int i = origin_y; i <= top_y; i++) {
        data_index = interpolation(rows, radius, i);
        input = (float4*)((uchar*)src + data_index * src_stride);
        for (int j = origin_x; j <= top_x; j++) {
          data_index = interpolation(cols, radius, j);
          value = input[data_index];
          if ((!unchecked0) && max.x == value.x) unchecked0 = false;
          if ((!unchecked1) && max.y == value.y) unchecked1 = false;
          if ((!unchecked2) && max.z == value.z) unchecked2 = false;
          if ((!unchecked3) && max.w == value.w) unchecked3 = false;

          if (unchecked0 && max.x == value.x) local_count.x++;
          if (unchecked1 && max.y == value.y) local_count.y++;
          if (unchecked2 && max.z == value.z) local_count.z++;
          if (unchecked3 && max.w == value.w) local_count.w++;

          if (index + global_count.x <= median_index && max.x < value.x &&
              value.x < top.x) {
            max.x = value.x;
            local_count.x = 0;
          }
          if (index + global_count.y <= median_index && max.y < value.y &&
              value.y < top.y) {
            max.y = value.y;
            local_count.y = 0;
          }
          if (index + global_count.z <= median_index && max.z < value.z &&
              value.z < top.z) {
            max.z = value.z;
            local_count.z = 0;
          }
          if (index + global_count.w <= median_index && max.w < value.w &&
              value.w < top.w) {
            max.w = value.w;
            local_count.w = 0;
          }
        }
      }
      global_count.x += local_count.x;
      global_count.y += local_count.y;
      global_count.z += local_count.z;
      global_count.w += local_count.w;
      if (max.x != FLT_MIN) top.x = max.x;
      if (max.y != FLT_MIN) top.y = max.y;
      if (max.z != FLT_MIN) top.z = max.z;
      if (max.w != FLT_MIN) top.w = max.w;
    }
  }

  float4* output = (float4*)((uchar*)dst + element_y * dst_stride);
  output[element_x] = top;
}

#define RUN_CHANNEL1_SMALL_KERNELS(Interpolation)                              \
Interpolation interpolation;                                                   \
medianC1SharedKernel<Interpolation><<<grid, block, 0, stream>>>(src, rows,     \
    cols, src_stride, median_index, radius, dst, dst_stride, interpolation);

#define RUN_CHANNELN_SMALL_KERNELS(Interpolation)                              \
Interpolation interpolation;                                                   \
if (channels == 3) {                                                           \
  medianC3SharedKernel<Interpolation><<<grid, block, 0, stream>>>(src, rows,   \
      cols, src_stride, median_index, radius, dst, dst_stride, interpolation); \
}                                                                              \
else {                                                                         \
  medianC4SharedKernel<Interpolation><<<grid, block, 0, stream>>>(src, rows,   \
      cols, src_stride, median_index, radius, dst, dst_stride, interpolation); \
}

#define RUN_KERNELS0(grid_x, Interpolation)                                    \
Interpolation interpolation;                                                   \
if (channels == 1) {                                                           \
  grid0.x = grid_x;                                                            \
  medianC1Kernel<Interpolation><<<grid0, block0, 0, stream>>>(src, rows, cols, \
      src_stride, median_index, radius, dst, dst_stride, interpolation);       \
}                                                                              \
else if (channels == 3) {                                                      \
  medianC3Kernel<Interpolation><<<grid0, block0, 0, stream>>>(src, rows, cols, \
      src_stride, median_index, radius, dst, dst_stride, interpolation);       \
}                                                                              \
else {                                                                         \
  medianC4Kernel<Interpolation><<<grid0, block0, 0, stream>>>(src, rows, cols, \
      src_stride, median_index, radius, dst, dst_stride, interpolation);       \
}

#define RUN_KERNELS1(grid_x, Interpolation)                                    \
Interpolation interpolation;                                                   \
if (channels == 1) {                                                           \
  grid0.x = grid_x;                                                            \
  medianC1Kernel<Interpolation><<<grid0, block0, 0, stream>>>(src, rows, cols, \
      src_stride, median_index, radius, dst, dst_stride, interpolation);       \
}                                                                              \
else if (channels == 3) {                                                      \
  medianC3Kernel<Interpolation><<<grid0, block0, 0, stream>>>(src, rows, cols, \
      src_stride, median_index, radius, dst, dst_stride, interpolation);       \
}                                                                              \
else {                                                                         \
  medianC4Kernel<Interpolation><<<grid0, block0, 0, stream>>>(src, rows, cols, \
      src_stride, median_index, radius, dst, dst_stride, interpolation);       \
}

RetCode medainblur(const uchar* src, int rows, int cols, int channels,
                   int src_stride, uchar* dst, int dst_stride, int ksize,
                   BorderType border_type, hipStream_t stream) {
  PPL_ASSERT(src != nullptr);
  PPL_ASSERT(dst != nullptr);
  PPL_ASSERT(rows >= 1 && cols >= 1);
  PPL_ASSERT(channels == 1 || channels == 3 || channels == 4);
  PPL_ASSERT(src_stride >= cols * channels * (int)sizeof(uchar));
  PPL_ASSERT(dst_stride >= cols * channels * (int)sizeof(uchar));
  PPL_ASSERT(ksize > 1);
  PPL_ASSERT((ksize & 1) == 1);
  PPL_ASSERT(border_type == BORDER_REPLICATE ||
             border_type == BORDER_REFLECT ||
             border_type == BORDER_REFLECT_101 ||
             border_type == BORDER_DEFAULT);

  uint radius = ksize >> 1;
  uint median_index = ksize * ksize >> 1;

  hipError_t code;
  if (ksize <= SMALL_KSIZE0 && channels == 1) {
    dim3 block, grid;
    block.x = kDimX0;
    block.y = kDimY0;
    grid.x = divideUp(divideUp(cols, 4, 2), kDimX0, kShiftX0);
    grid.y = divideUp(rows, kDimY0, kShiftY0);

    if (border_type == BORDER_REPLICATE) {
      RUN_CHANNEL1_SMALL_KERNELS(ReplicateBorder);
    }
    else if (border_type == BORDER_REFLECT) {
      RUN_CHANNEL1_SMALL_KERNELS(ReflectBorder);
    }
    else {
      RUN_CHANNEL1_SMALL_KERNELS(Reflect101Border);
    }

    code = hipGetLastError();
    if (code != hipSuccess) {
      LOG(ERROR) << "CUDA error: " << hipGetErrorString(code);
      return RC_DEVICE_RUNTIME_ERROR;
    }

    return RC_SUCCESS;
  }

  if (ksize <= SMALL_KSIZE1 && (channels == 3 || channels == 4)) {
    dim3 block, grid;
    block.x = kDimX0;
    block.y = kDimY0;
    grid.x = divideUp(cols, kDimX0, kShiftX0);
    grid.y = divideUp(rows, kDimY0, kShiftY0);

    if (border_type == BORDER_REPLICATE) {
      RUN_CHANNELN_SMALL_KERNELS(ReplicateBorder);
    }
    else if (border_type == BORDER_REFLECT) {
      RUN_CHANNELN_SMALL_KERNELS(ReflectBorder);
    }
    else {
      RUN_CHANNELN_SMALL_KERNELS(Reflect101Border);
    }

    code = hipGetLastError();
    if (code != hipSuccess) {
      LOG(ERROR) << "CUDA error: " << hipGetErrorString(code);
      return RC_DEVICE_RUNTIME_ERROR;
    }

    return RC_SUCCESS;
  }

  dim3 block0, grid0;
  block0.x = kBlockDimX0;
  block0.y = kBlockDimY0;
  grid0.x  = divideUp(cols, kBlockDimX0, kBlockShiftX0);
  grid0.y  = divideUp(rows, kBlockDimY0, kBlockShiftY0);

  int grid_x = divideUp(divideUp(cols, 4, 2), kBlockDimX0, kBlockShiftX0);
  if (border_type == BORDER_REPLICATE) {
    RUN_KERNELS0(grid_x, ReplicateBorder);
  }
  else if (border_type == BORDER_REFLECT) {
    RUN_KERNELS0(grid_x, ReflectBorder);
  }
  else {
    RUN_KERNELS0(grid_x, Reflect101Border);
  }

  code = hipGetLastError();
  if (code != hipSuccess) {
    LOG(ERROR) << "CUDA error: " << hipGetErrorString(code);
    return RC_DEVICE_RUNTIME_ERROR;
  }

  return RC_SUCCESS;
}

RetCode medainblur(const float* src, int rows, int cols, int channels,
                   int src_stride, float* dst, int dst_stride, int ksize,
                   BorderType border_type, hipStream_t stream) {
  PPL_ASSERT(src != nullptr);
  PPL_ASSERT(dst != nullptr);
  PPL_ASSERT(rows >= 1 && cols >= 1);
  PPL_ASSERT(channels == 1 || channels == 3 || channels == 4);
  PPL_ASSERT(src_stride >= cols * channels * (int)sizeof(float));
  PPL_ASSERT(dst_stride >= cols * channels * (int)sizeof(float));
  PPL_ASSERT(ksize > 1);
  PPL_ASSERT((ksize & 1) == 1);
  PPL_ASSERT(border_type == BORDER_REPLICATE ||
             border_type == BORDER_REFLECT ||
             border_type == BORDER_REFLECT_101 ||
             border_type == BORDER_DEFAULT);

  uint radius = ksize >> 1;
  uint median_index = ksize * ksize >> 1;

  hipError_t code;
  if (ksize <= SMALL_KSIZE0 && channels == 1) {
    dim3 block, grid;
    block.x = kDimX0;
    block.y = kDimY0;
    grid.x = divideUp(divideUp(cols, 4, 2), kDimX0, kShiftX0);
    grid.y = divideUp(rows, kDimY0, kShiftY0);

    if (border_type == BORDER_REPLICATE) {
      RUN_CHANNEL1_SMALL_KERNELS(ReplicateBorder);
    }
    else if (border_type == BORDER_REFLECT) {
      RUN_CHANNEL1_SMALL_KERNELS(ReflectBorder);
    }
    else {
      RUN_CHANNEL1_SMALL_KERNELS(Reflect101Border);
    }

    code = hipGetLastError();
    if (code != hipSuccess) {
      LOG(ERROR) << "CUDA error: " << hipGetErrorString(code);
      return RC_DEVICE_RUNTIME_ERROR;
    }

    return RC_SUCCESS;
  }

  if (ksize <= SMALL_KSIZE1 && (channels == 3 || channels == 4)) {
    dim3 block, grid;
    block.x = kDimX0;
    block.y = kDimY0;
    grid.x = divideUp(cols, kDimX0, kShiftX0);
    grid.y = divideUp(rows, kDimY0, kShiftY0);

    if (border_type == BORDER_REPLICATE) {
      RUN_CHANNELN_SMALL_KERNELS(ReplicateBorder);
    }
    else if (border_type == BORDER_REFLECT) {
      RUN_CHANNELN_SMALL_KERNELS(ReflectBorder);
    }
    else {
      RUN_CHANNELN_SMALL_KERNELS(Reflect101Border);
    }

    code = hipGetLastError();
    if (code != hipSuccess) {
      LOG(ERROR) << "CUDA error: " << hipGetErrorString(code);
      return RC_DEVICE_RUNTIME_ERROR;
    }

    return RC_SUCCESS;
  }

  dim3 block0, grid0;
  block0.x = kBlockDimX1;
  block0.y = kBlockDimY1;
  grid0.x  = divideUp(cols, kBlockDimX1, kBlockShiftX1);
  grid0.y  = divideUp(rows, kBlockDimY1, kBlockShiftY1);

  int grid_x = divideUp(divideUp(cols, 4, 2), kBlockDimX1, kBlockShiftX1);
  if (border_type == BORDER_REPLICATE) {
    RUN_KERNELS1(grid_x, ReplicateBorder);
  }
  else if (border_type == BORDER_REFLECT) {
    RUN_KERNELS1(grid_x, ReflectBorder);
  }
  else {
    RUN_KERNELS1(grid_x, Reflect101Border);
  }

  code = hipGetLastError();
  if (code != hipSuccess) {
    LOG(ERROR) << "CUDA error: " << hipGetErrorString(code);
    return RC_DEVICE_RUNTIME_ERROR;
  }

  return RC_SUCCESS;
}

template <>
RetCode MedianBlur<uchar, 1>(hipStream_t stream,
                             int height,
                             int width,
                             int inWidthStride,
                             const uchar* inData,
                             int outWidthStride,
                             uchar* outData,
                             int ksize,
                             BorderType border_type) {
  RetCode code = medainblur(inData, height, width, 1, inWidthStride, outData,
                            outWidthStride, ksize, border_type, stream);

  return code;
}

template <>
RetCode MedianBlur<uchar, 3>(hipStream_t stream,
                             int height,
                             int width,
                             int inWidthStride,
                             const uchar* inData,
                             int outWidthStride,
                             uchar* outData,
                             int ksize,
                             BorderType border_type) {
  RetCode code = medainblur(inData, height, width, 3, inWidthStride, outData,
                            outWidthStride, ksize, border_type, stream);

  return code;
}

template <>
RetCode MedianBlur<uchar, 4>(hipStream_t stream,
                             int height,
                             int width,
                             int inWidthStride,
                             const uchar* inData,
                             int outWidthStride,
                             uchar* outData,
                             int ksize,
                             BorderType border_type) {
  RetCode code = medainblur(inData, height, width, 4, inWidthStride, outData,
                            outWidthStride, ksize, border_type, stream);

  return code;
}

template <>
RetCode MedianBlur<float, 1>(hipStream_t stream,
                             int height,
                             int width,
                             int inWidthStride,
                             const float* inData,
                             int outWidthStride,
                             float* outData,
                             int ksize,
                             BorderType border_type) {
  inWidthStride  *= sizeof(float);
  outWidthStride *= sizeof(float);
  RetCode code = medainblur(inData, height, width, 1, inWidthStride, outData,
                            outWidthStride, ksize, border_type, stream);

  return code;
}

template <>
RetCode MedianBlur<float, 3>(hipStream_t stream,
                             int height,
                             int width,
                             int inWidthStride,
                             const float* inData,
                             int outWidthStride,
                             float* outData,
                             int ksize,
                             BorderType border_type) {
  inWidthStride  *= sizeof(float);
  outWidthStride *= sizeof(float);
  RetCode code = medainblur(inData, height, width, 3, inWidthStride, outData,
                            outWidthStride, ksize, border_type, stream);

  return code;
}

template <>
RetCode MedianBlur<float, 4>(hipStream_t stream,
                             int height,
                             int width,
                             int inWidthStride,
                             const float* inData,
                             int outWidthStride,
                             float* outData,
                             int ksize,
                             BorderType border_type) {
  inWidthStride  *= sizeof(float);
  outWidthStride *= sizeof(float);
  RetCode code = medainblur(inData, height, width, 4, inWidthStride, outData,
                            outWidthStride, ksize, border_type, stream);

  return code;
}

}  // cuda
}  // cv
}  // ppl
